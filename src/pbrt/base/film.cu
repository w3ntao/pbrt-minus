#include "hip/hip_runtime.h"
#include <ext/lodepng/lodepng.h>
#include <pbrt/base/film.h>
#include <pbrt/film/rgb_film.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/spectrum_util/color_encoding.h>
#include <vector>

Film *Film::create_rgb_film(const Filter *filter, const ParameterDictionary &parameters,
                            GPUMemoryAllocator &allocator) {
    auto rgb_film = RGBFilm::create(filter, parameters, allocator);

    auto film = allocator.allocate<Film>();
    film->init(rgb_film);

    return film;
}

void Film::init(RGBFilm *rgb_film) {
    ptr = rgb_film;
    type = Type::rgb;
}

PBRT_CPU_GPU
Point2i Film::get_resolution() const {
    switch (type) {
    case Type::rgb: {
        return static_cast<const RGBFilm *>(ptr)->get_resolution();
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_CPU_GPU
const Filter *Film::get_filter() const {
    switch (type) {
    case Type::rgb: {
        return static_cast<const RGBFilm *>(ptr)->get_filter();
    }
    }

    REPORT_FATAL_ERROR();
    return nullptr;
}

PBRT_CPU_GPU
Bounds2f Film::sample_bounds() const {
    switch (type) {
    case Type::rgb: {
        return static_cast<const RGBFilm *>(ptr)->sample_bounds();
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_CPU_GPU
void Film::add_sample(uint pixel_index, const SampledSpectrum &radiance_l,
                      const SampledWavelengths &lambda, Real weight) {
    switch (type) {
    case Type::rgb: {
        return static_cast<RGBFilm *>(ptr)->add_sample(pixel_index, radiance_l, lambda, weight);
    }
    }

    REPORT_FATAL_ERROR();
}

PBRT_CPU_GPU
void Film::add_sample(const Point2i &p_film, const SampledSpectrum &radiance_l,
                      const SampledWavelengths &lambda, Real weight) {
    switch (type) {
    case Type::rgb: {
        return static_cast<RGBFilm *>(ptr)->add_sample(p_film, radiance_l, lambda, weight);
    }
    }

    REPORT_FATAL_ERROR();
}

void Film::add_splat(const Point2f &p_film, const SampledSpectrum &radiance_l,
                     const SampledWavelengths &lambda) {
    switch (type) {
    case Type::rgb: {
        return static_cast<RGBFilm *>(ptr)->add_splat(p_film, radiance_l, lambda);
    }
    }

    REPORT_FATAL_ERROR();
}

PBRT_CPU_GPU
RGB Film::get_pixel_rgb(const Point2i &p, Real splat_scale) const {
    switch (type) {
    case Type::rgb: {
        return static_cast<RGBFilm *>(ptr)->get_pixel_rgb(p, splat_scale);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

__global__ void copy_pixels(uint8_t *gpu_frame_buffer, const Film *film, uint width, uint height,
                            Real splat_scale) {
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const auto worker_idx = y * width + x;

    auto rgb = film->get_pixel_rgb(Point2i(x, y), splat_scale);
    if (rgb.has_nan()) {
        gpu_frame_buffer[worker_idx * 3 + 0] = 0;
        gpu_frame_buffer[worker_idx * 3 + 1] = 0;
        gpu_frame_buffer[worker_idx * 3 + 2] = 0;

        return;
    }

    const SRGBColorEncoding srgb_encoding;

    gpu_frame_buffer[worker_idx * 3 + 0] = srgb_encoding.from_linear(rgb.r);
    gpu_frame_buffer[worker_idx * 3 + 1] = srgb_encoding.from_linear(rgb.g);
    gpu_frame_buffer[worker_idx * 3 + 2] = srgb_encoding.from_linear(rgb.b);
}

void Film::copy_to_frame_buffer(uint8_t *gpu_frame_buffer, Real splat_scale) const {
    const auto image_resolution = this->get_resolution();

    constexpr uint thread_width = 16;
    constexpr uint thread_height = 16;

    dim3 blocks(divide_and_ceil(uint(image_resolution.x), thread_width),
                divide_and_ceil(uint(image_resolution.y), thread_height));
    dim3 threads(thread_width, thread_height);

    copy_pixels<<<blocks, threads>>>(gpu_frame_buffer, this, image_resolution.x, image_resolution.y,
                                     splat_scale);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

void Film::write_to_png(const std::string &filename, Real splat_scale) const {
    auto resolution = get_resolution();

    int width = resolution.x;
    int height = resolution.y;

    SRGBColorEncoding srgb_encoding;
    std::vector<unsigned char> png_pixels(width * height * 4);

    uint nan_pixels = 0;

    for (uint y = 0; y < height; y++) {
        for (uint x = 0; x < width; x++) {
            uint index = y * width + x;
            auto rgb = get_pixel_rgb(Point2i(x, y), splat_scale);
            if (rgb.has_nan()) {
                nan_pixels += 1;
            }

            png_pixels[4 * index + 0] = srgb_encoding.from_linear(rgb.r);
            png_pixels[4 * index + 1] = srgb_encoding.from_linear(rgb.g);
            png_pixels[4 * index + 2] = srgb_encoding.from_linear(rgb.b);
            png_pixels[4 * index + 3] = 255;
        }
    }

    if (nan_pixels > 0) {
        printf("%sFilm::%s(): %d/%d (%.2f%) pixels with NAN component%s\n",
               FLAG_COLORFUL_PRINT_RED_START, __func__, nan_pixels, width * height,
               double(nan_pixels) / (width * height) * 100, FLAG_COLORFUL_PRINT_END);
    }

    // Encode the image
    // if there's an error, display it
    if (unsigned error = lodepng::encode(filename, png_pixels, width, height); error) {
        std::cerr << "lodepng::encoder error " << error << ": " << lodepng_error_text(error)
                  << std::endl;
        throw std::runtime_error("lodepng::encode() fail");
    }
}
