#include "hip/hip_runtime.h"
#include <pbrt/accelerator/hlbvh.h>
#include <pbrt/base/film.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/light.h>
#include <pbrt/base/material.h>
#include <pbrt/base/sampler.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/gui/gl_helper.h>
#include <pbrt/integrators/wavefront_path.h>
#include <pbrt/light_samplers/power_light_sampler.h>
#include <pbrt/samplers/independent.h>
#include <pbrt/samplers/stratified.h>
#include <pbrt/scene/parameter_dictionary.h>
#include <pbrt/spectrum_util/sampled_spectrum.h>
#include <pbrt/spectrum_util/sampled_wavelengths.h>
#include <pbrt/util/basic_math.h>

constexpr uint PATH_POOL_SIZE = 2 * 1024 * 1024;

struct FrameBuffer {
    uint pixel_idx;
    uint sample_idx;
    SampledSpectrum radiance;
    SampledWavelengths lambda;
    FloatType weight;

    // to help sorting
    bool operator<(const FrameBuffer &right) const {
        if (pixel_idx < right.pixel_idx) {
            return true;
        }

        if (pixel_idx > right.pixel_idx) {
            return false;
        }

        return sample_idx < right.sample_idx;
    }
};

struct MISParameter {
    bool specular_bounce = true;
    bool any_non_specular_bounces = false;

    FloatType pdf_bsdf;
    FloatType eta_scale;
    LightSampleContext prev_interaction_light_sample_ctx;

    PBRT_CPU_GPU
    void init() {
        specular_bounce = true;
        any_non_specular_bounces = false;

        pdf_bsdf = NAN;
        eta_scale = 1.0;
    }
};

static __global__ void gpu_init_independent_samplers(Sampler *samplers,
                                                     IndependentSampler *independent_samplers,
                                                     uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    samplers[worker_idx].init(&independent_samplers[worker_idx]);
}

static __global__ void gpu_init_stratified_samplers(Sampler *samplers,
                                                    StratifiedSampler *stratified_samplers,
                                                    uint samples_per_dimension, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    stratified_samplers[worker_idx].init(samples_per_dimension);

    samplers[worker_idx].init(&stratified_samplers[worker_idx]);
}

static __global__ void gpu_init_path_state(WavefrontPathIntegrator::PathState *path_state) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }

    path_state->init_new_path(worker_idx);
}

__global__ void control_logic(const WavefrontPathIntegrator *integrator,
                              WavefrontPathIntegrator::PathState *path_state,
                              WavefrontPathIntegrator::Queues *queues) {
    const uint path_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_idx >= PATH_POOL_SIZE || path_state->finished[path_idx]) {
        return;
    }

    // otherwise beta is larger than 0.0
    auto &isect = path_state->shape_intersections[path_idx]->interaction;
    const auto ray = path_state->camera_rays[path_idx].ray;
    auto &lambda = path_state->lambdas[path_idx];

    const auto path_length = path_state->path_length[path_idx];
    const auto specular_bounce = path_state->mis_parameters[path_idx].specular_bounce;
    auto &beta = path_state->beta[path_idx];
    auto &L = path_state->L[path_idx];

    const auto prev_interaction_light_sample_ctx =
        path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx;
    const auto pdf_bsdf = path_state->mis_parameters[path_idx].pdf_bsdf;

    bool should_terminate_path = !path_state->shape_intersections[path_idx].has_value() ||
                                 path_length > integrator->max_depth || !beta.is_positive();

    if (!should_terminate_path && path_length > 8) {
        // possibly terminate the path with Russian roulette

        auto &eta_scale = path_state->mis_parameters[path_idx].eta_scale;
        auto &sampler = path_state->samplers[path_idx];
        const auto u = sampler.get_1d();
        // consume this random value anyway to keep samples aligned

        SampledSpectrum russian_roulette_beta = beta * eta_scale;
        if (russian_roulette_beta.max_component_value() < 1) {
            auto q = clamp<FloatType>(1 - russian_roulette_beta.max_component_value(), 0, 0.95);
            if (u < q) {
                beta = SampledSpectrum(0.0);
                should_terminate_path = true;
            } else {
                beta /= 1 - q;
            }
        }
    }

    if (should_terminate_path) {
        if (beta.is_positive()) {
            // sample infinite lights
            for (uint idx = 0; idx < integrator->base->infinite_light_num; ++idx) {
                auto light = integrator->base->infinite_lights[idx];
                auto Le = light->le(ray, lambda);

                if (path_length == 0 || specular_bounce) {
                    L += beta * Le;
                } else {
                    // Compute MIS weight for infinite light
                    FloatType pdf_light =
                        integrator->base->light_sampler->pmf(prev_interaction_light_sample_ctx,
                                                             light) *
                        light->pdf_li(prev_interaction_light_sample_ctx, ray.d, true);
                    FloatType weight_bsdf = power_heuristic(1, pdf_bsdf, 1, pdf_light);

                    L += beta * weight_bsdf * Le;
                }
            }
        }

        const uint queue_idx = atomicAdd(&queues->frame_buffer_counter, 1);
        queues->frame_buffer_queue[queue_idx] = FrameBuffer{
            .pixel_idx = path_state->pixel_indices[path_idx],
            .sample_idx = path_state->sample_indices[path_idx],
            .radiance = L * path_state->camera_rays[path_idx].weight,
            .lambda = lambda,
            .weight = path_state->camera_samples[path_idx].filter_weight,
        };

        queues->new_path_queue[atomicAdd(&queues->new_path_counter, 1)] = path_idx;
        return;
    }

    SampledSpectrum Le = isect.le(-ray.d, lambda);
    if (Le.is_positive()) {
        if (path_length == 0 || specular_bounce)
            path_state->L[path_idx] += beta * Le;
        else {
            // Compute MIS weight for area light
            auto area_light = isect.area_light;

            FloatType pdf_light = integrator->base->light_sampler->pmf(
                                      prev_interaction_light_sample_ctx, area_light) *
                                  area_light->pdf_li(prev_interaction_light_sample_ctx, ray.d);
            FloatType weight_light = power_heuristic(1, pdf_bsdf, 1, pdf_light);

            path_state->L[path_idx] += beta * weight_light * Le;
        }
    }

    // for active paths: advance one segment

    path_state->path_length[path_idx] += 1;

    switch (isect.material->get_material_type()) {

    case Material::Type::conductor: {
        const uint queue_idx = atomicAdd(&queues->conductor_material_counter, 1);
        queues->conductor_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::coated_conductor: {
        const uint queue_idx = atomicAdd(&queues->coated_conductor_material_counter, 1);
        queues->coated_conductor_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::coated_diffuse: {
        const uint queue_idx = atomicAdd(&queues->coated_diffuse_material_counter, 1);
        queues->coated_diffuse_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::dielectric: {
        const uint queue_idx = atomicAdd(&queues->dielectric_material_counter, 1);
        queues->dielectric_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::diffuse: {
        const uint queue_idx = atomicAdd(&queues->diffuse_material_counter, 1);
        queues->diffuse_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::mix: {
        printf("\nyou should not see MixMaterial here\n\n");
        REPORT_FATAL_ERROR();
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }
}

__global__ void write_frame_buffer(Film *film, WavefrontPathIntegrator::Queues *queues) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->frame_buffer_counter) {
        return;
    }

    const auto pixel_idx = queues->frame_buffer_queue[queue_idx].pixel_idx;
    if (queue_idx > 0 && pixel_idx == queues->frame_buffer_queue[queue_idx - 1].pixel_idx) {
        return;
    }

    for (uint idx = queue_idx; idx < queues->frame_buffer_counter &&
                               queues->frame_buffer_queue[idx].pixel_idx == pixel_idx;
         ++idx) {
        // make sure the same pixels are written by the same thread
        const auto &frame_buffer = queues->frame_buffer_queue[idx];
        film->add_sample(frame_buffer.pixel_idx, frame_buffer.radiance, frame_buffer.lambda,
                         frame_buffer.weight);
    }
}

__global__ void fill_new_path_queue(WavefrontPathIntegrator::Queues *queues) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }
    queues->new_path_queue[worker_idx] = worker_idx;
}

__global__ void generate_new_path(const IntegratorBase *base,
                                  WavefrontPathIntegrator::PathState *path_state,
                                  WavefrontPathIntegrator::Queues *queues) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->new_path_counter) {
        return;
    }

    const uint path_idx = queues->new_path_queue[queue_idx];

    const auto unique_path_id = atomicAdd(&path_state->global_path_counter, 1);
    if (unique_path_id >= path_state->total_path_num) {
        path_state->finished[path_idx] = true;
        return;
    }

    const uint width = path_state->image_resolution.x;
    const uint height = path_state->image_resolution.y;

    const uint pixel_idx = unique_path_id % (width * height);
    const uint sample_idx = unique_path_id / (width * height);

    auto sampler = &path_state->samplers[path_idx];

    auto p_pixel = Point2i(pixel_idx % width, pixel_idx / width);

    sampler->start_pixel_sample(pixel_idx, sample_idx, 0);

    path_state->camera_samples[path_idx] = sampler->get_camera_sample(p_pixel, base->filter);
    auto lu = sampler->get_1d();
    path_state->lambdas[path_idx] = SampledWavelengths::sample_visible(lu);

    path_state->camera_rays[path_idx] =
        base->camera->generate_ray(path_state->camera_samples[path_idx], sampler);

    path_state->pixel_indices[path_idx] = pixel_idx;
    path_state->sample_indices[path_idx] = sample_idx;
    path_state->path_length[path_idx] = 0;

    path_state->init_new_path(path_idx);

    uint ray_queue_idx = atomicAdd(&queues->ray_counter, 1);
    queues->ray_queue[ray_queue_idx] = path_idx;
}

__global__ void gpu_evaluate_material(WavefrontPathIntegrator::PathState *path_state,
                                      WavefrontPathIntegrator::Queues *queues,
                                      const Material::Type material_type,
                                      const WavefrontPathIntegrator *integrator) {
    uint material_counter = 0;
    uint *material_queue = nullptr;

    switch (material_type) {
    case Material::Type::coated_conductor: {
        material_counter = queues->coated_conductor_material_counter;
        material_queue = queues->coated_conductor_material_queue;
        break;
    }

    case Material::Type::coated_diffuse: {
        material_counter = queues->coated_diffuse_material_counter;
        material_queue = queues->coated_diffuse_material_queue;
        break;
    }

    case Material::Type::conductor: {
        material_counter = queues->conductor_material_counter;
        material_queue = queues->conductor_material_queue;
        break;
    }

    case Material::Type::dielectric: {
        material_counter = queues->dielectric_material_counter;
        material_queue = queues->dielectric_material_queue;
        break;
    }

    case Material::Type::diffuse: {
        material_counter = queues->diffuse_material_counter;
        material_queue = queues->diffuse_material_queue;
        break;
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }

    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= material_counter) {
        return;
    }

    const uint path_idx = material_queue[queue_idx];

    auto &lambda = path_state->lambdas[path_idx];

    auto sampler = &path_state->samplers[path_idx];

    auto &isect = path_state->shape_intersections[path_idx]->interaction;

    path_state->bsdf[path_idx] =
        isect.get_bsdf(lambda, integrator->base->camera, sampler->get_samples_per_pixel());

    integrator->sample_bsdf(path_idx, path_state);

    uint ray_queue_idx = atomicAdd(&queues->ray_counter, 1);
    queues->ray_queue[ray_queue_idx] = path_idx;
}

__global__ void ray_cast(const WavefrontPathIntegrator *integrator,
                         WavefrontPathIntegrator::PathState *path_state,
                         WavefrontPathIntegrator::Queues *queues) {
    const uint ray_queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_queue_idx >= queues->ray_counter) {
        return;
    }

    const uint path_idx = queues->ray_queue[ray_queue_idx];

    const auto camera_ray = path_state->camera_rays[path_idx];

    path_state->shape_intersections[path_idx] =
        integrator->base->intersect(camera_ray.ray, Infinity);
}

PBRT_CPU_GPU
void WavefrontPathIntegrator::sample_bsdf(uint path_idx, PathState *path_state) const {
    auto &isect = path_state->shape_intersections[path_idx]->interaction;
    auto &lambda = path_state->lambdas[path_idx];

    auto &ray = path_state->camera_rays[path_idx].ray;
    auto sampler = &path_state->samplers[path_idx];

    if (regularize && path_state->mis_parameters[path_idx].any_non_specular_bounces) {
        path_state->bsdf[path_idx].regularize();
    }

    if (pbrt::is_non_specular(path_state->bsdf[path_idx].flags())) {
        SampledSpectrum Ld = sample_ld(isect, &path_state->bsdf[path_idx], lambda, sampler);
        path_state->L[path_idx] += path_state->beta[path_idx] * Ld;
    }

    // Sample BSDF to get new path direction
    Vector3f wo = -ray.d;
    FloatType u = sampler->get_1d();
    auto bs = path_state->bsdf[path_idx].sample_f(wo, u, sampler->get_2d());
    if (!bs) {
        path_state->beta[path_idx] = SampledSpectrum(0.0);
        return;
    }

    path_state->beta[path_idx] *= bs->f * bs->wi.abs_dot(isect.shading.n.to_vector3()) / bs->pdf;

    path_state->mis_parameters[path_idx].pdf_bsdf =
        bs->pdf_is_proportional ? path_state->bsdf[path_idx].pdf(wo, bs->wi) : bs->pdf;
    path_state->mis_parameters[path_idx].specular_bounce = bs->is_specular();
    path_state->mis_parameters[path_idx].any_non_specular_bounces |= (!bs->is_specular());

    if (bs->is_transmission()) {
        path_state->mis_parameters[path_idx].eta_scale *= sqr(bs->eta);
    }

    path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx = isect;

    path_state->camera_rays[path_idx].ray = isect.spawn_ray(bs->wi);
}

void WavefrontPathIntegrator::evaluate_material(const Material::Type material_type) {
    uint material_counter = 0;
    switch (material_type) {
    case Material::Type::coated_conductor: {
        material_counter = queues.coated_conductor_material_counter;
        break;
    }

    case Material::Type::coated_diffuse: {
        material_counter = queues.coated_diffuse_material_counter;
        break;
    }

    case Material::Type::conductor: {
        material_counter = queues.conductor_material_counter;
        break;
    }

    case Material::Type::dielectric: {
        material_counter = queues.dielectric_material_counter;
        break;
    }

    case Material::Type::diffuse: {
        material_counter = queues.diffuse_material_counter;
        break;
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }

    if (material_counter <= 0) {
        return;
    }

    const uint threads = 256;
    const auto blocks = divide_and_ceil(material_counter, threads);

    gpu_evaluate_material<<<blocks, threads>>>(&path_state, &queues, material_type, this);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

PBRT_CPU_GPU
void WavefrontPathIntegrator::PathState::init_new_path(uint path_idx) {
    finished[path_idx] = false;
    shape_intersections[path_idx].reset();

    L[path_idx] = SampledSpectrum(0.0);
    beta[path_idx] = SampledSpectrum(1.0);
    path_length[path_idx] = 0;

    mis_parameters[path_idx].init();
}

void WavefrontPathIntegrator::PathState::create(uint samples_per_pixel, const Point2i &_resolution,
                                                const std::string &sampler_type,
                                                GPUMemoryAllocator &allocator) {
    image_resolution = _resolution;
    global_path_counter = 0;
    total_path_num = samples_per_pixel * image_resolution.x * image_resolution.y;

    camera_samples = allocator.allocate<CameraSample>(PATH_POOL_SIZE);
    camera_rays = allocator.allocate<CameraRay>(PATH_POOL_SIZE);
    lambdas = allocator.allocate<SampledWavelengths>(PATH_POOL_SIZE);

    L = allocator.allocate<SampledSpectrum>(PATH_POOL_SIZE);
    beta = allocator.allocate<SampledSpectrum>(PATH_POOL_SIZE);
    shape_intersections = allocator.allocate<pbrt::optional<ShapeIntersection>>(PATH_POOL_SIZE);

    path_length = allocator.allocate<uint>(PATH_POOL_SIZE);
    finished = allocator.allocate<bool>(PATH_POOL_SIZE);
    pixel_indices = allocator.allocate<uint>(PATH_POOL_SIZE);
    sample_indices = allocator.allocate<uint>(PATH_POOL_SIZE);

    bsdf = allocator.allocate<BSDF>(PATH_POOL_SIZE);
    mis_parameters = allocator.allocate<MISParameter>(PATH_POOL_SIZE);
    samplers = allocator.allocate<Sampler>(PATH_POOL_SIZE);

    constexpr uint threads = 1024;
    uint blocks = divide_and_ceil<uint>(PATH_POOL_SIZE, threads);

    if (sampler_type == "stratified") {
        const auto samples_per_dimension = static_cast<int>(std::sqrt(samples_per_pixel));
        if (samples_per_dimension * samples_per_dimension != samples_per_pixel) {
            REPORT_FATAL_ERROR();
        }

        auto stratified_samplers = allocator.allocate<StratifiedSampler>(PATH_POOL_SIZE);

        gpu_init_stratified_samplers<<<blocks, threads>>>(samplers, stratified_samplers,
                                                          samples_per_dimension, PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else if (sampler_type == "independent") {
        auto independent_samplers = allocator.allocate<IndependentSampler>(PATH_POOL_SIZE);

        gpu_init_independent_samplers<<<blocks, threads>>>(samplers, independent_samplers,
                                                           PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else {
        REPORT_FATAL_ERROR();
    }

    gpu_init_path_state<<<PATH_POOL_SIZE, threads>>>(this);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

void WavefrontPathIntegrator::Queues::init(GPUMemoryAllocator &allocator) {
    new_path_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    ray_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    frame_buffer_queue = allocator.allocate<FrameBuffer>(PATH_POOL_SIZE);

    coated_conductor_material_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    coated_diffuse_material_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    conductor_material_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    dielectric_material_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
    diffuse_material_queue = allocator.allocate<uint>(PATH_POOL_SIZE);
}

WavefrontPathIntegrator *WavefrontPathIntegrator::create(uint samples_per_pixel,
                                                         const std::string &sampler_type,
                                                         const ParameterDictionary &parameters,
                                                         const IntegratorBase *base,
                                                         GPUMemoryAllocator &allocator) {
    auto integrator = allocator.allocate<WavefrontPathIntegrator>();

    integrator->samples_per_pixel = samples_per_pixel;

    integrator->base = base;
    integrator->path_state.create(samples_per_pixel, base->camera->get_camerabase()->resolution,
                                  sampler_type, allocator);

    integrator->queues.init(allocator);

    integrator->max_depth = parameters.get_integer("maxdepth", 5);
    integrator->regularize = parameters.get_bool("regularize", false);

    return integrator;
}

PBRT_CPU_GPU
SampledSpectrum WavefrontPathIntegrator::sample_ld(const SurfaceInteraction &intr, const BSDF *bsdf,
                                                   SampledWavelengths &lambda,
                                                   Sampler *sampler) const {
    // Initialize _LightSampleContext_ for light sampling
    LightSampleContext ctx(intr);
    // Try to nudge the light sampling position to correct side of the surface
    BxDFFlags flags = bsdf->flags();
    if (pbrt::is_reflective(flags) && !pbrt::is_transmissive(flags)) {
        ctx.pi = intr.offset_ray_origin(intr.wo);
    } else if (pbrt::is_transmissive(flags) && !pbrt::is_reflective(flags)) {
        ctx.pi = intr.offset_ray_origin(-intr.wo);
    }

    // Choose a light source for the direct lighting calculation
    FloatType u = sampler->get_1d();
    auto sampled_light = base->light_sampler->sample(ctx, u);

    Point2f uLight = sampler->get_2d();
    if (!sampled_light) {
        return SampledSpectrum(0);
    }

    // Sample a point on the light source for direct lighting
    auto light = sampled_light->light;
    auto ls = light->sample_li(ctx, uLight, lambda);
    if (!ls || !ls->l.is_positive() || ls->pdf == 0) {
        return SampledSpectrum(0);
    }

    // Evaluate BSDF for light sample and check light visibility
    Vector3f wo = intr.wo;
    Vector3f wi = ls->wi;
    SampledSpectrum f = bsdf->f(wo, wi) * wi.abs_dot(intr.shading.n.to_vector3());

    if (!f.is_positive() || !base->unoccluded(intr, ls->p_light)) {
        return SampledSpectrum(0);
    }

    // Return light's contribution to reflected radiance
    FloatType pdf_light = sampled_light->p * ls->pdf;
    if (pbrt::is_delta_light(light->get_light_type())) {
        return ls->l * f / pdf_light;
    }

    // for non delta light
    FloatType pdf_bsdf = bsdf->pdf(wo, wi);
    FloatType weight_light = power_heuristic(1, pdf_light, 1, pdf_bsdf);

    return weight_light * ls->l * f / pdf_light;
}

void WavefrontPathIntegrator::render(Film *film, const bool preview) {
    printf("wavefront: path pool size: %u\n", PATH_POOL_SIZE);

    const auto image_resolution = this->path_state.image_resolution;

    const auto num_pixels = image_resolution.x * image_resolution.y;

    GLHelper gl_helper;
    if (preview) {
        gl_helper.init("initializing", image_resolution);
    }

    constexpr uint threads = 256;

    // generate new paths for the whole pool
    fill_new_path_queue<<<PATH_POOL_SIZE, threads>>>(&queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    queues.new_path_counter = PATH_POOL_SIZE;

    queues.ray_counter = 0;
    generate_new_path<<<divide_and_ceil(queues.new_path_counter, threads), threads>>>(
        base, &path_state, &queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    while (queues.ray_counter > 0) {
        ray_cast<<<divide_and_ceil(queues.ray_counter, threads), threads>>>(this, &path_state,
                                                                            &queues);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // clear all queues before control stage
        queues.new_path_counter = 0;
        queues.ray_counter = 0;
        queues.frame_buffer_counter = 0;

        queues.coated_conductor_material_counter = 0;
        queues.coated_diffuse_material_counter = 0;
        queues.conductor_material_counter = 0;
        queues.dielectric_material_counter = 0;
        queues.diffuse_material_counter = 0;

        control_logic<<<divide_and_ceil(PATH_POOL_SIZE, threads), threads>>>(this, &path_state,
                                                                             &queues);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        if (queues.frame_buffer_counter > 0) {
            // sort to make film writing deterministic
            std::sort(queues.frame_buffer_queue + 0,
                      queues.frame_buffer_queue + queues.frame_buffer_counter, std::less{});

            write_frame_buffer<<<divide_and_ceil(queues.frame_buffer_counter, threads), threads>>>(
                film, &queues);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());

            if (preview) {
                film->copy_to_frame_buffer(gl_helper.gpu_frame_buffer);

                const auto current_sample_idx =
                    std::min<uint>(path_state.global_path_counter / num_pixels, samples_per_pixel);

                gl_helper.draw_frame(
                    GLHelper::assemble_title(FloatType(current_sample_idx) / samples_per_pixel));
            }
        }

        if (queues.new_path_counter > 0) {
            generate_new_path<<<divide_and_ceil(queues.new_path_counter, threads), threads>>>(
                base, &path_state, &queues);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }

        for (const auto material_type : {
                 Material::Type::coated_conductor,
                 Material::Type::coated_diffuse,
                 Material::Type::conductor,
                 Material::Type::dielectric,
                 Material::Type::diffuse,
             }) {
            evaluate_material(material_type);
        }
    }
}
