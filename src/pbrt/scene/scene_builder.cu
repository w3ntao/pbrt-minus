#include "hip/hip_runtime.h"
#include <pbrt/accelerator/hlbvh.h>
#include <pbrt/base/film.h>
#include <pbrt/base/filter.h>
#include <pbrt/base/float_texture.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/material.h>
#include <pbrt/base/megakernel_integrator.h>
#include <pbrt/base/primitive.h>
#include <pbrt/base/sampler.h>
#include <pbrt/base/shape.h>
#include <pbrt/films/grey_scale_film.h>
#include <pbrt/integrators/bdpt.h>
#include <pbrt/integrators/mlt_path.h>
#include <pbrt/integrators/wavefront_path.h>
#include <pbrt/light_samplers/power_light_sampler.h>
#include <pbrt/light_samplers/uniform_light_sampler.h>
#include <pbrt/scene/scene_builder.h>
#include <pbrt/spectrum_util/global_spectra.h>
#include <pbrt/spectrum_util/spectrum_constants_glass.h>
#include <pbrt/spectrum_util/spectrum_constants_metal.h>
#include <pbrt/textures/spectrum_constant_texture.h>
#include <pbrt/util/std_container.h>
#include <set>

uint next_keyword_position(const std::vector<Token> &tokens, uint start) {
    for (uint idx = start + 1; idx < tokens.size(); ++idx) {
        const auto type = tokens[idx].type;
        if (type == TokenType::Number || type == TokenType::String || type == TokenType::Variable ||
            type == TokenType::List) {
            continue;
        }
        return idx;
    }

    return tokens.size();
}

void add_one_to_map(const std::string &key, std::map<std::string, uint> &counter) {
    if (counter.find(key) == counter.end()) {
        counter[key] = 1;
        return;
    }

    counter[key] += 1;
}

std::map<std::string, uint> count_light_type(const std::vector<Light *> &gpu_lights) {
    std::map<std::string, uint> counter;
    for (const auto light : gpu_lights) {
        switch (light->type) {
        case Light::Type::diffuse_area_light: {
            add_one_to_map("DiffuseAreaLight", counter);
            break;
        }

        case Light::Type::distant_light: {
            add_one_to_map("DistantLight", counter);
            break;
        }

        case Light::Type::image_infinite_light: {
            add_one_to_map("ImageInfiniteLight", counter);
            break;
        }

        case Light::Type::spot_light: {
            add_one_to_map("SpotLight", counter);
            break;
        }

        case Light::Type::uniform_infinite_light: {
            add_one_to_map("UniformInfiniteLight", counter);
            break;
        }

        default: {
            REPORT_FATAL_ERROR();
        }
        }
    }

    return counter;
}

std::map<std::string, uint> count_material_type(const std::vector<const Primitive *> &primitives) {
    const std::map<Material::Type, std::string> material_names = {
        {Material::Type::coated_conductor, "CoatedConductor"},
        {Material::Type::coated_diffuse, "CoatedDiffuse"},
        {Material::Type::conductor, "Conductor"},
        {Material::Type::dielectric, "Dielectric"},
        {Material::Type::diffuse, "Diffuse"},
        {Material::Type::diffuse_transmission, "DiffuseTransmission"},
        {Material::Type::mix, "Mix"},
    };

    std::map<std::string, uint> counter;
    for (const auto primitive : primitives) {
        const auto type = primitive->get_material()->get_material_type();
        if (material_names.find(type) == material_names.end()) {
            REPORT_FATAL_ERROR();
        }

        const auto name = material_names.at(type);
        add_one_to_map(name, counter);
    }

    return counter;
}

SceneBuilder::SceneBuilder(const CommandLineOption &command_line_option)
    : integrator_name(command_line_option.integrator_name),
      output_filename(command_line_option.output_file),
      samples_per_pixel(command_line_option.samples_per_pixel),
      preview(command_line_option.preview) {

    global_spectra = GlobalSpectra::create(RGBtoSpectrumData::Gamut::sRGB, allocator);

    auto ag_eta = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Ag_eta, false,
                                                                              nullptr, allocator);

    auto ag_k = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Ag_k, false, nullptr,
                                                                            allocator);

    auto al_eta = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Al_eta, false,
                                                                              nullptr, allocator);

    auto al_k = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Al_k, false, nullptr,
                                                                            allocator);

    auto au_eta = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Au_eta, false,
                                                                              nullptr, allocator);

    auto au_k = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Au_k, false, nullptr,
                                                                            allocator);

    auto cu_eta = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Cu_eta, false,
                                                                              nullptr, allocator);

    auto cu_k = Spectrum::create_piecewise_linear_spectrum_from_interleaved(Cu_k, false, nullptr,
                                                                            allocator);

    auto glass_bk7_eta = Spectrum::create_piecewise_linear_spectrum_from_interleaved(
        GlassBK7_eta, false, nullptr, allocator);

    spectra = {
        {"metal-Ag-eta", ag_eta},     {"metal-Ag-k", ag_k},     {"metal-Al-eta", al_eta},
        {"metal-Al-k", al_k},         {"metal-Au-eta", au_eta}, {"metal-Au-k", au_k},
        {"metal-Cu-eta", cu_eta},     {"metal-Cu-k", cu_k},

        {"glass-BK7", glass_bk7_eta},
    };

    integrator_base = allocator.allocate<IntegratorBase>();
    integrator_base->init();

    auto texture = SpectrumTexture::create_constant_float_val_texture(0.5, allocator);
    graphics_state.material = Material::create_diffuse_material(texture, allocator);
}

void SceneBuilder::build_camera() {
    if (film == nullptr) {
        REPORT_FATAL_ERROR();
    }

    const auto parameters = build_parameter_dictionary(sub_vector(camera_tokens, 2));

    const auto camera_type = camera_tokens[1].values[0];
    if (camera_type == "perspective") {
        auto camera_from_world = graphics_state.transform;
        auto world_from_camera = camera_from_world.inverse();

        named_coordinate_systems["camera"] = world_from_camera;

        auto camera_transform =
            CameraTransform(world_from_camera, RenderingCoordinateSystem::CameraWorldCoordSystem);

        render_from_world = camera_transform.render_from_world;

        if (this->film == nullptr || integrator_base->filter == nullptr) {
            REPORT_FATAL_ERROR();
        }

        integrator_base->camera =
            Camera::create_perspective_camera(film->get_resolution(), camera_transform, this->film,
                                              integrator_base->filter, parameters, allocator);

        return;
    }

    printf("\n%s(): Camera type `%s` not implemented\n", __func__, camera_type.c_str());
    REPORT_FATAL_ERROR();
}

void SceneBuilder::build_filter() {
    ParameterDictionary parameters;
    std::string filter_type = "mitchell";
    if (!pixel_filter_tokens.empty()) {
        parameters = build_parameter_dictionary(sub_vector(pixel_filter_tokens, 2));
        filter_type = pixel_filter_tokens[1].values[0];
    }

    integrator_base->filter = Filter::create(filter_type, parameters, allocator);
}

void SceneBuilder::build_film() {
    const auto parameters = build_parameter_dictionary(sub_vector(film_tokens, 2));

    if (output_filename.empty()) {
        output_filename = parameters.get_one_string("filename");
    }

    if (std::filesystem::path p(output_filename); p.extension() != ".png") {
        printf("output filename extension: only PNG is supported for the moment\n");
        output_filename = p.replace_extension(".png").filename();
    }

    if (integrator_base->filter == nullptr) {
        REPORT_FATAL_ERROR();
    }
    film = Film::create_rgb_film(integrator_base->filter, parameters, allocator);
}

void SceneBuilder::build_gpu_lights() {
    auto light_array = allocator.allocate<const Light *>(gpu_lights.size());
    CHECK_CUDA_ERROR(hipMemcpy(light_array, gpu_lights.data(), sizeof(Light *) * gpu_lights.size(),
                                hipMemcpyHostToDevice));

    integrator_base->lights = light_array;
    integrator_base->light_num = gpu_lights.size();

    integrator_base->light_sampler =
        PowerLightSampler::create(light_array, gpu_lights.size(), allocator);

    std::vector<const Light *> infinite_lights;
    for (auto light : gpu_lights) {
        if (light->get_light_type() == LightType::infinite) {
            infinite_lights.push_back(light);
        }
    }

    auto gpu_infinite_lights = allocator.allocate<const Light *>(infinite_lights.size());

    CHECK_CUDA_ERROR(hipMemcpy(gpu_infinite_lights, infinite_lights.data(),
                                sizeof(Light *) * infinite_lights.size(), hipMemcpyHostToDevice));

    integrator_base->infinite_lights = gpu_infinite_lights;
    integrator_base->infinite_light_num = infinite_lights.size();
}

void SceneBuilder::build_integrator() {
    build_gpu_lights();

    const auto parameters = build_parameter_dictionary(sub_vector(integrator_tokens, 2));

    if (!integrator_name.has_value()) {
        integrator_name = parameters.get_one_string("Integrator", "path");
    }

    if (!samples_per_pixel.has_value()) {
        samples_per_pixel = 4;
    }

    const std::string sampler_type = "stratified";
    // const std::string sampler_type = "independent";

    if (sampler_type == "stratified" && integrator_name->find("mlt") == std::string::npos) {
        // MLT integrator ues it's own sampler
        const auto sqrt_val = int(std::sqrt(samples_per_pixel.value()));
        samples_per_pixel = sqr(sqrt_val);
    }

    if (integrator_name == "volpath") {
        integrator_name = "path";
    }

    if (integrator_name == "bdpt") {
        bdpt_integrator = BDPTIntegrator::create(samples_per_pixel.value(), sampler_type,
                                                 parameters, integrator_base, allocator);
        return;
    }

    if (integrator_name == "mlt" || integrator_name == "mltpath") {
        mlt_integrator = MLTPathIntegrator::create(samples_per_pixel.value(), parameters,
                                                   integrator_base, allocator);
        return;
    }

    printf("sampler: %s\n", sampler_type.c_str());

    if (integrator_name == "path") {
        wavefront_path_integrator = WavefrontPathIntegrator::create(
            samples_per_pixel.value(), sampler_type, parameters, integrator_base, allocator);
        return;
    }

    megakernel_integrator = MegakernelIntegrator::create(integrator_name.value(), parameters,
                                                         integrator_base, allocator);
}

void SceneBuilder::parse_keyword(const std::vector<Token> &tokens) {
    const auto keyword = tokens[0].values[0];

    if (keyword == "AreaLightSource") {
        parse_area_light_source(tokens);
        return;
    }

    if (keyword == "Camera") {
        camera_tokens = tokens;

        auto camera_from_world = graphics_state.transform;
        named_coordinate_systems["camera"] = camera_from_world.inverse();

        return;
    }

    if (keyword == "ConcatTransform") {
        parse_concat_transform(tokens);
        return;
    }

    if (keyword == "CoordSysTransform") {
        auto coord_sys_name = tokens[1].values[0];
        if (named_coordinate_systems.find(coord_sys_name) == named_coordinate_systems.end()) {
            printf("\ncoordinate system `%s` not available\n", coord_sys_name.c_str());
            REPORT_FATAL_ERROR();
        }

        graphics_state.transform = named_coordinate_systems.at(coord_sys_name);
        return;
    }

    if (keyword == "Film") {
        film_tokens = tokens;
        return;
    }

    if (keyword == "Identity") {
        graphics_state.transform = Transform::identity();
        return;
    }

    if (keyword == "Include") {
        auto included_file = tokens[1].values[0];
        parse_file(get_file_full_path(included_file));
        return;
    }

    if (keyword == "Integrator") {
        if (integrator_name.has_value()) {
            // ignore config file, when integrator is read from command line option
            return;
        }

        integrator_name = tokens[1].values[0];
        integrator_tokens = tokens;
        return;
    }

    if (keyword == "LightSource") {
        parse_light_source(tokens);
        return;
    }

    if (keyword == "LookAt") {
        parse_lookat(tokens);
        return;
    }

    if (keyword == "MakeNamedMaterial") {
        parse_make_named_material(tokens);
        return;
    }

    if (keyword == "Material") {
        parse_material(tokens);
        return;
    }

    if (keyword == "NamedMaterial") {
        parse_named_material(tokens);
        return;
    }

    if (keyword == "PixelFilter") {
        pixel_filter_tokens = tokens;
        return;
    }

    if (keyword == "ReverseOrientation") {
        graphics_state.reverse_orientation = !graphics_state.reverse_orientation;
        return;
    }

    if (keyword == "Rotate") {
        parse_rotate(tokens);
        return;
    }

    if (keyword == "Sampler") {
        const auto parameters = build_parameter_dictionary(sub_vector(tokens, 2));

        if (!samples_per_pixel.has_value()) {
            samples_per_pixel = parameters.get_integer("pixelsamples", 4);
        }

        return;
    }

    if (keyword == "Scale") {
        parse_scale(tokens);
        return;
    }

    if (keyword == "Shape") {
        parse_shape(tokens);
        return;
    }

    if (keyword == "Texture") {
        parse_texture(tokens);
        return;
    }

    if (keyword == "Transform") {
        parse_transform(tokens);
        return;
    }

    if (keyword == "Translate") {
        parse_translate(tokens);
        return;
    }

    if (keyword == "MakeNamedMedium" || keyword == "MediumInterface") {

        static std::set<std::string> unimplemented_keywords;
        if (unimplemented_keywords.find(keyword) == unimplemented_keywords.end()) {
            unimplemented_keywords.insert(keyword);
            printf("%s(): keyword `%s` not implemented\n", __func__, keyword.c_str());
        }

        return;
    }

    printf("\n%s(): `%s` not implemented\n", __func__, keyword.c_str());
    REPORT_FATAL_ERROR();
}

void SceneBuilder::parse_concat_transform(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "ConcatTransform")) {
        REPORT_FATAL_ERROR();
    }

    std::vector<FloatType> data(16);
    for (uint idx = 0; idx < tokens[1].values.size(); idx++) {
        data[idx] = stod(tokens[1].values[idx]);
    }

    FloatType transform_data[4][4];
    for (uint y = 0; y < 4; y++) {
        for (uint x = 0; x < 4; x++) {
            transform_data[y][x] = data[y * 4 + x];
        }
    }

    auto transform_matrix = SquareMatrix<4>(transform_data);

    graphics_state.transform = graphics_state.transform * transform_matrix.transpose();
}

void SceneBuilder::parse_light_source(const std::vector<Token> &tokens) {
    const auto parameters = build_parameter_dictionary(sub_vector(tokens, 2));

    const auto light_source_type = tokens[1].values[0];

    auto light = Light::create(light_source_type, get_render_from_object(), parameters, allocator);
    gpu_lights.push_back(light);
}

void SceneBuilder::parse_lookat(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "LookAt")) {
        throw std::runtime_error("expect Keyword(LookAt)");
    }

    std::vector<FloatType> data;
    for (int idx = 1; idx < tokens.size(); idx++) {
        data.push_back(tokens[idx].to_float());
    }

    auto position = Point3f(data[0], data[1], data[2]);
    auto look = Point3f(data[3], data[4], data[5]);
    auto up = Vector3f(data[6], data[7], data[8]);

    graphics_state.transform = graphics_state.transform * Transform::lookat(position, look, up);
}

void SceneBuilder::parse_make_named_material(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "MakeNamedMaterial")) {
        REPORT_FATAL_ERROR();
    }

    const auto material_name = tokens[1].values[0];

    const auto parameters = build_parameter_dictionary(sub_vector(tokens, 2));

    auto type_of_material = parameters.get_one_string("type");

    materials[material_name] = Material::create(type_of_material, parameters, allocator);
}

void SceneBuilder::parse_material(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "Material")) {
        REPORT_FATAL_ERROR();
    }

    auto type_of_material = tokens[1].values[0];

    const auto parameters = build_parameter_dictionary(sub_vector(tokens, 2));

    graphics_state.material = Material::create(type_of_material, parameters, allocator);
}

void SceneBuilder::parse_named_material(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "NamedMaterial")) {
        REPORT_FATAL_ERROR();
    }

    const auto material_name = tokens[1].values[0];

    if (materials.find(material_name) == materials.end()) {
        REPORT_FATAL_ERROR();
    }

    graphics_state.material = materials.at(material_name);
}

void SceneBuilder::parse_rotate(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "Rotate")) {
        REPORT_FATAL_ERROR();
    }

    std::vector<FloatType> data;
    for (int idx = 1; idx < tokens.size(); idx++) {
        data.push_back(tokens[idx].to_float());
    }

    graphics_state.transform =
        graphics_state.transform * Transform::rotate(data[0], data[1], data[2], data[3]);
}

void SceneBuilder::parse_scale(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "Scale")) {
        REPORT_FATAL_ERROR();
    }

    std::vector<FloatType> data;
    for (int idx = 1; idx < tokens.size(); idx++) {
        data.push_back(tokens[idx].to_float());
    }

    graphics_state.transform *= Transform::scale(data[0], data[1], data[2]);
}

void SceneBuilder::parse_area_light_source(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "AreaLightSource")) {
        REPORT_FATAL_ERROR();
    }

    if (tokens[1] != Token(TokenType::String, "diffuse")) {
        throw std::runtime_error("parse_area_light_source: only `diffuse` supported at the moment");
    }

    graphics_state.area_light_entity =
        AreaLightEntity(tokens[1].values[0], build_parameter_dictionary(sub_vector(tokens, 2)));
}

void SceneBuilder::parse_shape(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "Shape")) {
        REPORT_FATAL_ERROR();
    }

    const auto parameters = build_parameter_dictionary(sub_vector(tokens, 2));
    auto type_of_shape = tokens[1].values[0];
    const auto render_from_object = get_render_from_object();

    auto result = Shape::create(type_of_shape, render_from_object, render_from_object.inverse(),
                                graphics_state.reverse_orientation, parameters, allocator);
    auto shapes = result.first;
    auto num_shapes = result.second;

    if (!graphics_state.area_light_entity) {
        auto simple_primitives = Primitive::create_simple_primitives(
            shapes, graphics_state.material, num_shapes, allocator);

        if (active_instance_definition) {
            active_instance_definition->instantiated_primitives.push_back(
                InstantiatedPrimitive(simple_primitives, num_shapes));
        } else {
            for (uint idx = 0; idx < num_shapes; ++idx) {
                gpu_primitives.push_back(&simple_primitives[idx]);
            }
        }

        return;
    }

    if (active_instance_definition) {
        printf("\nERROR: area lights not supported with object instancing\n");
        REPORT_FATAL_ERROR();
    }

    auto diffuse_area_lights =
        Light::create_diffuse_area_lights(shapes, num_shapes, render_from_object,
                                          graphics_state.area_light_entity->parameters, allocator);

    auto geometric_primitives = Primitive::create_geometric_primitives(
        shapes, graphics_state.material, diffuse_area_lights, num_shapes, allocator);

    // otherwise: build AreaDiffuseLight
    for (uint idx = 0; idx < num_shapes; ++idx) {
        auto primitive_ptr = &geometric_primitives[idx];
        auto area_light_ptr = &diffuse_area_lights[idx];

        gpu_lights.push_back(area_light_ptr);
        gpu_primitives.push_back(primitive_ptr);
    }
}

void SceneBuilder::parse_texture(const std::vector<Token> &tokens) {
    auto texture_name = tokens[1].values[0];
    auto color_type = tokens[2].values[0];
    auto texture_type = tokens[3].values[0];
    const auto parameters = build_parameter_dictionary(sub_vector(tokens, 4));

    if (color_type == "float") {
        auto float_texture =
            FloatTexture::create(texture_type, get_render_from_object(), parameters, allocator);
        float_textures[texture_name] = float_texture;

        return;
    }

    if (color_type == "spectrum") {
        albedo_spectrum_textures[texture_name] =
            SpectrumTexture::create(texture_type, SpectrumType::Albedo, get_render_from_object(),
                                    global_spectra->rgb_color_space, parameters, allocator);

        illuminant_spectrum_textures[texture_name] = SpectrumTexture::create(
            texture_type, SpectrumType::Illuminant, get_render_from_object(),
            global_spectra->rgb_color_space, parameters, allocator);

        unbounded_spectrum_textures[texture_name] =
            SpectrumTexture::create(texture_type, SpectrumType::Unbounded, get_render_from_object(),
                                    global_spectra->rgb_color_space, parameters, allocator);

        return;
    }

    printf("\n%s(): color type `%s` not implemented\n", __func__, color_type.c_str());
    REPORT_FATAL_ERROR();
}

void SceneBuilder::parse_transform(const std::vector<Token> &tokens) {
    if (tokens[0] != Token(TokenType::Keyword, "Transform")) {
        REPORT_FATAL_ERROR();
    }

    std::vector<FloatType> data(16);
    for (uint idx = 0; idx < tokens[1].values.size(); idx++) {
        data[idx] = stod(tokens[1].values[idx]);
    }

    FloatType transform_data[4][4];
    for (uint y = 0; y < 4; y++) {
        for (uint x = 0; x < 4; x++) {
            transform_data[y][x] = data[y * 4 + x];
        }
    }

    const auto transform_matrix = SquareMatrix<4>(transform_data);

    graphics_state.transform = transform_matrix.transpose();
}

void SceneBuilder::parse_translate(const std::vector<Token> &tokens) {
    std::vector<FloatType> data;
    for (int idx = 1; idx < tokens.size(); idx++) {
        data.push_back(tokens[idx].to_float());
    }

    graphics_state.transform *= Transform::translate(data[0], data[1], data[2]);
}

void SceneBuilder::parse_tokens(const std::vector<Token> &tokens) {
    uint token_idx = 0;
    while (token_idx < tokens.size()) {
        const Token &first_token = tokens[token_idx];
        if (first_token.type == TokenType::WorldBegin) {
            build_filter();
            build_film();
            build_camera();

            graphics_state.transform = Transform::identity();
            named_coordinate_systems["world"] = graphics_state.transform;

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::AttributeBegin) {
            pushed_graphics_state.push(graphics_state);

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::AttributeEnd) {
            if (pushed_graphics_state.empty()) {
                REPORT_FATAL_ERROR();
            }

            graphics_state = pushed_graphics_state.top();
            pushed_graphics_state.pop();

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::ObjectBegin) {
            pushed_graphics_state.push(graphics_state);

            if (active_instance_definition) {
                printf("\nERROR: ObjectBegin called inside of instance definition\n");
                REPORT_FATAL_ERROR();
            }

            active_instance_definition = std::make_shared<ActiveInstanceDefinition>();

            active_instance_definition->name = first_token.values[0];

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::ObjectEnd) {
            if (!active_instance_definition) {
                printf("\nERROR: ObjectEnd called before an instance defined\n");
                REPORT_FATAL_ERROR();
            }

            instance_definition[active_instance_definition->name] = active_instance_definition;

            active_instance_definition = nullptr;

            graphics_state = pushed_graphics_state.top();
            pushed_graphics_state.pop();

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::ObjectInstance) {
            const auto object_name = first_token.values[0];
            if (instance_definition.find(object_name) == instance_definition.end()) {
                printf("\nERROR: object `%s` not found\n", object_name.c_str());
                REPORT_FATAL_ERROR();
            }

            const auto instance = instance_definition.at(object_name);

            auto world_from_render = render_from_world.inverse();
            auto render_from_instance = get_render_from_object() * world_from_render;

            if (render_from_instance.is_identity()) {
                for (auto &instanced_primitives : instance->instantiated_primitives) {
                    for (uint p_idx = 0; p_idx < instanced_primitives.num; ++p_idx) {
                        gpu_primitives.push_back(&instanced_primitives.primitives[p_idx]);
                    }
                }
            } else {
                for (auto &instanced_primitives : instance->instantiated_primitives) {
                    auto transformed_primitives = Primitive::create_transformed_primitives(
                        instanced_primitives.primitives, render_from_instance,
                        instanced_primitives.num, allocator);

                    for (uint p_idx = 0; p_idx < instanced_primitives.num; ++p_idx) {
                        gpu_primitives.push_back(&transformed_primitives[p_idx]);
                    }
                }
            }

            token_idx += 1;
            continue;
        }

        if (first_token.type == TokenType::Keyword) {
            auto end = next_keyword_position(tokens, token_idx);
            auto keyword_tokens = sub_vector(tokens, token_idx, end);

            parse_keyword(keyword_tokens);

            token_idx = end;
            continue;
        }

        std::cout << "\nillegal token: \n" << first_token << "\n";
        REPORT_FATAL_ERROR();
    }
}

void SceneBuilder::parse_file(const std::string &_filename) {
    const auto all_tokens = parse_pbrt_into_token(_filename);
    parse_tokens(all_tokens);
}

void SceneBuilder::preprocess() {
    integrator_base->bvh = HLBVH::create(gpu_primitives, allocator);

    const auto full_scene_bounds = integrator_base->bvh->bounds();
    for (auto light : gpu_lights) {
        light->preprocess(full_scene_bounds);
    }

    build_integrator();

    if (bdpt_integrator != nullptr) {
        printf("Integrator: (wavefront) bdpt\n");
    } else if (mlt_integrator != nullptr) {
        printf("Integrator: (wavefront) mlt\n");
    } else if (wavefront_path_integrator != nullptr) {
        printf("Integrator: (wavefront) path\n");
    } else if (megakernel_integrator != nullptr) {
        printf("Integrator: (megakernel) %s\n", megakernel_integrator->get_name().c_str());
    } else {
        REPORT_FATAL_ERROR();
    }
    printf("\n");

    auto light_type_counter = count_light_type(gpu_lights);

    auto light_size = gpu_lights.size();
    printf("total lights: %zu\n", light_size);
    for (auto const &kv : light_type_counter) {
        printf("    %s: %d (%.2f%)\n", kv.first.c_str(), kv.second,
               double(kv.second) / light_size * 100);
    }
    printf("\n");

    auto primitives_size = gpu_primitives.size();
    auto material_type_counter = count_material_type(gpu_primitives);
    printf("materials' type: %zu\n", material_type_counter.size());
    for (auto const &kv : material_type_counter) {
        printf("    %s: %d (%.2f%)\n", kv.first.c_str(), kv.second,
               double(kv.second) / primitives_size * 100);
    }
    printf("\n");
}

void SceneBuilder::render() const {
    if (!integrator_base->is_ready()) {
        REPORT_FATAL_ERROR();
    }

    if (!samples_per_pixel.has_value()) {
        REPORT_FATAL_ERROR();
    }

    if (film == nullptr) {
        REPORT_FATAL_ERROR();
    }

    const auto film_resolution = film->get_resolution();

    std::string sampler_type = "stratified";
    // TODO: configure sampler_type

    auto start = std::chrono::system_clock::now();

    std::cout << "rendering a " << film_resolution.x << "x" << film_resolution.y << " image";

    const auto spp = samples_per_pixel.value();

    if (bdpt_integrator != nullptr) {
        std::cout << " (samples per pixel: " << spp << ")"
                  << " with BDPT\n"
                  << std::flush;

        auto splat_scale = 1.0 / spp;

        bdpt_integrator->render(film, spp, preview);

        film->write_to_png(output_filename, splat_scale);

    } else if (mlt_integrator != nullptr) {
        std::cout << " (mutations per pixel: " << spp << ")"
                  << " with MLT-path\n"
                  << std::flush;

        GreyScaleFilm heatmap(film_resolution);

        const auto brightness = mlt_integrator->render(film, heatmap, spp, preview);

        film->write_to_png(output_filename, brightness / spp);

        heatmap.write_to_png("heatmap-" + output_filename);

    } else if (wavefront_path_integrator != nullptr) {
        std::cout << " (samples per pixel: " << spp << ")"
                  << " with wavefront-path\n"
                  << std::flush;

        wavefront_path_integrator->render(film, preview);

        film->write_to_png(output_filename);

    } else if (megakernel_integrator != nullptr) {
        std::cout << " (samples per pixel: " << spp << ")"
                  << " with " + megakernel_integrator->get_name() << "\n"
                  << std::flush;

        megakernel_integrator->render(film, sampler_type, samples_per_pixel.value(),
                                      integrator_base, preview);

        film->write_to_png(output_filename);

    } else {
        REPORT_FATAL_ERROR();
    }

    const std::chrono::duration<FloatType> duration{std::chrono::system_clock::now() - start};

    std::cout << std::fixed << std::setprecision(1) << "rendering took " << duration.count()
              << " seconds\n"
              << std::flush;

    printf("GPU memory used: %s\n", allocator.get_allocated_memory_size().c_str());

    std::cout << "image saved to `" << output_filename << "`\n";
}
