#include "hip/hip_runtime.h"
#include <pbrt/accelerator/hlbvh.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/sampler.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/integrators/megakernel_path.h>
#include <pbrt/light_samplers/power_light_sampler.h>
#include <pbrt/lights/image_infinite_light.h>
#include <pbrt/medium/homogeneous_medium.h>
#include <pbrt/scene/parameter_dictionary.h>
#include <pbrt/spectra/densely_sampled_spectrum.h>

const MegakernelPathIntegrator *
MegakernelPathIntegrator::create(const ParameterDictionary &parameters,
                                 const IntegratorBase *integrator_base,
                                 GPUMemoryAllocator &allocator) {
    auto max_depth = parameters.get_integer("maxdepth", 5);
    auto regularize = parameters.get_bool("regularize", false);

    auto path_integrator = allocator.allocate<MegakernelPathIntegrator>();
    *path_integrator = MegakernelPathIntegrator(integrator_base, max_depth, regularize);

    return path_integrator;
}

constexpr int rr_depth = 8;                         // TODO: move rr_depth into namespace pbrt
constexpr Real russian_roulette_upper_bound = 0.95; // TODO: move rr_upper into namespace pbrt

PBRT_CPU_GPU
static bool should_terminate(const int bounces, const int max_depth, SampledSpectrum &throughput,
                             Sampler *sampler) {
    if (bounces >= max_depth) {
        return true;
    }

    if (bounces >= rr_depth) {
        const auto survive_prob =
            std::fmin(throughput.max_component_value(), russian_roulette_upper_bound);
        if (sampler->get_1d() > survive_prob) {
            return true;
        }

        throughput /= survive_prob;
    }

    return false;
}

PBRT_CPU_GPU
SampledSpectrum MegakernelPathIntegrator::evaluate_li_volume(const Ray &primary_ray,
                                                             SampledWavelengths &lambda,
                                                             const IntegratorBase *base,
                                                             Sampler *sampler, int max_depth,
                                                             bool regularize) {
    auto L = SampledSpectrum(0.0);
    auto beta = SampledSpectrum(1.0);
    bool specular_bounce = false;
    bool any_non_specular_bounces = false;

    pbrt::optional<Real> prev_direction_pdf;
    pbrt::optional<SurfaceInteraction> prev_interaction;
    Real multi_transmittance_pdf = 1.0;

    auto ray = primary_ray;

    int bounces = 0;
    while (true) {
        auto optional_intersection = base->intersect(ray, Infinity);
        if (ray.medium) {
            const SampledSpectrum sigma_a = ray.medium->sigma_a->sample(lambda);
            const SampledSpectrum sigma_s = ray.medium->sigma_s->sample(lambda);
            const SampledSpectrum sigma_t = sigma_a + sigma_s;
            const auto sigma_t_avg = sigma_t.average();

            const auto t_transmittance =
                optional_intersection ? optional_intersection->t_hit : Infinity;
            const auto u = sampler->get_1d();
            if (const auto t = -std::log(1 - u) / sigma_t_avg; t < t_transmittance) {
                ray.o = ray.at(t);
                beta /= sigma_t;

                SurfaceInteraction medium_interaction;
                medium_interaction.pi = ray.o;
                medium_interaction.wo = -ray.d;
                medium_interaction.medium = ray.medium;

                SampledSpectrum Ld =
                    sample_ld_volume(medium_interaction, nullptr, lambda, base, sampler);
                L += beta * Ld * sigma_s;

                auto phase_sample = ray.medium->phase.sample(-ray.d, sampler->get_2d());
                if (!phase_sample) {
                    break;
                }

                beta *= phase_sample->rho / phase_sample->pdf * sigma_s;

                if (should_terminate(bounces, max_depth, beta, sampler)) {
                    break;
                }
                bounces += 1;

                prev_direction_pdf = phase_sample->pdf;

                prev_interaction = medium_interaction;
                multi_transmittance_pdf = 1.0;
                ray.d = phase_sample->wi;
                continue;
            }

            auto transmittance_pdf = std::exp(-sigma_t_avg * t_transmittance);
            multi_transmittance_pdf *= transmittance_pdf;
        }

        if (!optional_intersection) {
            // Incorporate emission from infinite lights for escaped ray
            for (int idx = 0; idx < base->infinite_light_num; ++idx) {
                auto light = base->infinite_lights[idx];
                auto Le = light->le(ray, lambda);

                if (bounces == 0 || specular_bounce) {
                    L += beta * Le;
                } else {
                    // Compute MIS weight for infinite light
                    Real pdf_light = base->light_sampler->pmf(light) *
                                     light->pdf_li(*prev_interaction, ray.d, true);
                    Real dir_pdf = *prev_direction_pdf * multi_transmittance_pdf;

                    Real w = power_heuristic(1, dir_pdf, 1, pdf_light);

                    L += beta * w * Le;
                }
            }

            break;
        }

        SurfaceInteraction &surface_interaction = optional_intersection->interaction;
        if (!surface_interaction.material) {
            // pass through material-less interface
            ray = surface_interaction.spawn_ray(ray.d);
            continue;
        }

        // Incorporate emission from surface hit by ray
        if (const SampledSpectrum Le = surface_interaction.le(-ray.d, lambda); Le.is_positive()) {
            if (bounces == 0 || specular_bounce) {
                L += beta * Le;
            } else {
                // Compute MIS weight for area light
                auto area_light = surface_interaction.area_light;

                Real pdf_light = base->light_sampler->pmf(area_light) *
                                 area_light->pdf_li(*prev_interaction, ray.d);

                Real dir_pdf = *prev_direction_pdf * multi_transmittance_pdf;

                Real w = power_heuristic(1, dir_pdf, 1, pdf_light);

                L += beta * w * Le;
            }
        }

        auto bsdf =
            surface_interaction.get_bsdf(lambda, base->camera, sampler->get_samples_per_pixel());
        if (regularize && any_non_specular_bounces) {
            bsdf.regularize();
        }

        // Sample direct illumination from the light sources

        if (pbrt::is_non_specular(bsdf.flags())) {
            SampledSpectrum Ld =
                sample_ld_volume(surface_interaction, &bsdf, lambda, base, sampler);
            L += beta * Ld;
        }

        Vector3f wo = -ray.d;
        auto bs = bsdf.sample_f(wo, sampler->get_1d(), sampler->get_2d());
        if (!bs) {
            break;
        }

        beta *= bs->f * bs->wi.abs_dot(surface_interaction.shading.n.to_vector3()) / bs->pdf;

        if (should_terminate(bounces, max_depth, beta, sampler)) {
            break;
        }
        bounces += 1;

        prev_direction_pdf = bs->pdf_is_proportional ? bsdf.pdf(wo, bs->wi) : bs->pdf;
        multi_transmittance_pdf = 1.0;

        specular_bounce = bs->is_specular();
        any_non_specular_bounces |= !specular_bounce;

        prev_interaction = surface_interaction;
        ray = surface_interaction.spawn_ray(bs->wi);
        // different with PBRT-v4: ignore the DifferentialRay
    }

    return L;
}

PBRT_CPU_GPU
SampledSpectrum MegakernelPathIntegrator::li(const Ray &primary_ray, SampledWavelengths &lambda,
                                             Sampler *sampler) const {
    return evaluate_li_volume(primary_ray, lambda, base, sampler, max_depth, regularize);
}

PBRT_CPU_GPU
SampledSpectrum
MegakernelPathIntegrator::sample_ld_volume(const SurfaceInteraction &surface_interaction,
                                           const BSDF *bsdf, SampledWavelengths &lambda,
                                           const IntegratorBase *base, Sampler *sampler) {
    LightSampleContext ctx(surface_interaction);
    // Try to nudge the light sampling position to correct side of the surface
    if (bsdf) {
        const BxDFFlags flags = bsdf->flags();
        if (pbrt::is_reflective(flags) && !pbrt::is_transmissive(flags)) {
            ctx.pi = surface_interaction.offset_ray_origin(surface_interaction.wo);
        } else if (pbrt::is_transmissive(flags) && !pbrt::is_reflective(flags)) {
            ctx.pi = surface_interaction.offset_ray_origin(-surface_interaction.wo);
        }
    }

    // Choose a light source for the direct lighting calculation
    const Real u = sampler->get_1d();
    const Point2f u_light = sampler->get_2d();

    auto sampled_light = base->light_sampler->sample(ctx, u);
    if (!sampled_light) {
        return SampledSpectrum(0);
    }

    // Sample a point on the light source for direct lighting
    const auto light = sampled_light->light;
    const auto ls = light->sample_li(ctx, u_light, lambda);
    if (!ls || !ls->l.is_positive() || ls->pdf == 0) {
        return SampledSpectrum(0);
    }

    Real T_light = 1.0;
    Real pdf_transmittance_dir = 1.0; // for multiple importance sampling

    auto shadow_ray = surface_interaction.spawn_ray_to(ls->p_light, true);
    while (true) {
        const auto distance_to_light = ls->p_light.p().distance(shadow_ray.o);
        auto optional_intersection =
            base->intersect(shadow_ray, (1.0 - ShadowEpsilon) * distance_to_light);
        const auto next_t =
            optional_intersection ? optional_intersection->t_hit : distance_to_light;

        if (shadow_ray.medium) {
            const SampledSpectrum sigma_t = shadow_ray.medium->sample_sigma_t(lambda);
            const auto sigma_t_avg = sigma_t.average();

            T_light *= std::exp(-sigma_t_avg * next_t);
            pdf_transmittance_dir *= std::exp(-sigma_t_avg * next_t);
        }

        if (!optional_intersection) {
            // shadow ray hit nothing: reach light
            break;
        }

        // ray hit something in between light and origin
        if (optional_intersection->interaction.material) {
            // got blocked by some primitives
            return SampledSpectrum(0);
        }

        // otherwise hit material-less shape
        shadow_ray = optional_intersection->interaction.spawn_ray_to(ls->p_light, true);
    }

    const auto pdf_light = sampled_light->pdf * ls->pdf;
    const auto light_contribution = ls->l / pdf_light * T_light;

    if (bsdf) {
        // Evaluate BSDF for light sample and check light visibility
        Vector3f wo = surface_interaction.wo;
        Vector3f wi = ls->wi;
        SampledSpectrum f =
            bsdf->f(wo, wi) * wi.abs_dot(surface_interaction.shading.n.to_vector3());

        if (!f.is_positive() || !base->unoccluded(surface_interaction, ls->p_light)) {
            return SampledSpectrum(0);
        }

        // Return light's contribution to reflected radiance
        if (pbrt::is_delta_light(light->get_light_type())) {
            return f * light_contribution;
        }

        // for non delta light
        const auto pdf_bsdf = bsdf->pdf(wo, wi) * pdf_transmittance_dir;
        const auto w = power_heuristic(1, pdf_light, 1, pdf_bsdf);

        return w * f * light_contribution;
    }
    // else: sample Ld in volume

    auto phase_sample =
        surface_interaction.medium->phase.sample(surface_interaction.wo, sampler->get_2d());

    if (!phase_sample) {
        return SampledSpectrum(0);
    }

    if (pbrt::is_delta_light(light->get_light_type())) {
        // NO MIS for delta light
        return phase_sample->rho * light_contribution;
    }

    const auto pdf_phase = phase_sample->pdf * pdf_transmittance_dir;
    const auto w = power_heuristic(1, pdf_light, 1, pdf_phase);

    return w * phase_sample->rho * light_contribution;
}
