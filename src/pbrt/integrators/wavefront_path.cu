#include "hip/hip_runtime.h"
#include <pbrt/accelerator/hlbvh.h>
#include <pbrt/base/film.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/light.h>
#include <pbrt/base/material.h>
#include <pbrt/base/sampler.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/gui/gl_helper.h>
#include <pbrt/integrators/wavefront_path.h>
#include <pbrt/light_samplers/power_light_sampler.h>
#include <pbrt/samplers/independent.h>
#include <pbrt/samplers/stratified.h>
#include <pbrt/scene/parameter_dictionary.h>
#include <pbrt/spectrum_util/sampled_spectrum.h>
#include <pbrt/spectrum_util/sampled_wavelengths.h>
#include <pbrt/util/math.h>

constexpr uint PATH_POOL_SIZE = 2 * 1024 * 1024;

struct FrameBuffer {
    uint pixel_idx;
    uint sample_idx;
    SampledSpectrum radiance;
    SampledWavelengths lambda;
    FloatType weight;

    // to help sorting
    bool operator<(const FrameBuffer &right) const {
        if (pixel_idx < right.pixel_idx) {
            return true;
        }

        if (pixel_idx > right.pixel_idx) {
            return false;
        }

        return sample_idx < right.sample_idx;
    }
};

struct MISParameter {
    bool specular_bounce = true;
    bool any_non_specular_bounces = false;

    FloatType pdf_bsdf;
    FloatType eta_scale;
    LightSampleContext prev_interaction_light_sample_ctx;

    PBRT_CPU_GPU
    void init() {
        specular_bounce = true;
        any_non_specular_bounces = false;

        pdf_bsdf = NAN;
        eta_scale = 1.0;
    }
};

static __global__ void gpu_init_independent_samplers(Sampler *samplers,
                                                     IndependentSampler *independent_samplers,
                                                     uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    samplers[worker_idx].init(&independent_samplers[worker_idx]);
}

static __global__ void gpu_init_stratified_samplers(Sampler *samplers,
                                                    StratifiedSampler *stratified_samplers,
                                                    uint samples_per_dimension, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    stratified_samplers[worker_idx].init(samples_per_dimension);

    samplers[worker_idx].init(&stratified_samplers[worker_idx]);
}

static __global__ void gpu_init_path_state(WavefrontPathIntegrator::PathState *path_state) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }

    path_state->init_new_path(worker_idx);
}

__global__ void control_logic(WavefrontPathIntegrator::PathState *path_state,
                              WavefrontPathIntegrator::Queues *queues, const uint max_depth,
                              const IntegratorBase *base) {
    const uint path_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_idx >= PATH_POOL_SIZE || path_state->finished[path_idx]) {
        return;
    }

    auto &isect = path_state->shape_intersections[path_idx]->interaction;
    const auto ray = path_state->camera_rays[path_idx].ray;
    auto &lambda = path_state->lambdas[path_idx];

    const auto path_length = path_state->path_length[path_idx];
    const auto specular_bounce = path_state->mis_parameters[path_idx].specular_bounce;
    auto &beta = path_state->beta[path_idx];
    auto &L = path_state->L[path_idx];

    const auto prev_interaction_light_sample_ctx =
        path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx;
    const auto pdf_bsdf = path_state->mis_parameters[path_idx].pdf_bsdf;

    bool should_terminate_path = !path_state->shape_intersections[path_idx].has_value() ||
                                 path_length > max_depth || !beta.is_positive();

    if (!should_terminate_path && path_length > 8) {
        // possibly terminate the path with Russian roulette

        auto &eta_scale = path_state->mis_parameters[path_idx].eta_scale;
        auto &sampler = path_state->samplers[path_idx];
        const auto u = sampler.get_1d();
        // consume this random value anyway to keep samples aligned

        SampledSpectrum russian_roulette_beta = beta * eta_scale;
        if (russian_roulette_beta.max_component_value() < 1) {
            auto q = clamp<FloatType>(1 - russian_roulette_beta.max_component_value(), 0, 0.95);
            if (u < q) {
                beta = SampledSpectrum(0.0);
                should_terminate_path = true;
            } else {
                beta /= 1 - q;
            }
        }
    }

    if (should_terminate_path) {
        if (beta.is_positive()) {
            // sample infinite lights
            for (uint idx = 0; idx < base->infinite_light_num; ++idx) {
                auto light = base->infinite_lights[idx];
                auto Le = light->le(ray, lambda);

                if (path_length == 0 || specular_bounce) {
                    L += beta * Le;
                } else {
                    // Compute MIS weight for infinite light
                    FloatType pdf_light =
                        base->light_sampler->pmf(prev_interaction_light_sample_ctx, light) *
                        light->pdf_li(prev_interaction_light_sample_ctx, ray.d, true);
                    FloatType weight_bsdf = power_heuristic(1, pdf_bsdf, 1, pdf_light);

                    L += beta * weight_bsdf * Le;
                }
            }
        }

        const uint queue_idx = atomicAdd(&queues->frame_buffer_counter, 1);
        queues->frame_buffer_queue[queue_idx] = FrameBuffer{
            .pixel_idx = path_state->pixel_indices[path_idx],
            .sample_idx = path_state->sample_indices[path_idx],
            .radiance = L * path_state->camera_rays[path_idx].weight,
            .lambda = lambda,
            .weight = path_state->camera_samples[path_idx].filter_weight,
        };

        queues->new_paths->append_path(path_idx);
        return;
    }

    SampledSpectrum Le = isect.le(-ray.d, lambda);
    if (Le.is_positive()) {
        if (path_length == 0 || specular_bounce)
            path_state->L[path_idx] += beta * Le;
        else {
            // Compute MIS weight for area light
            auto area_light = isect.area_light;

            FloatType pdf_light =
                base->light_sampler->pmf(prev_interaction_light_sample_ctx, area_light) *
                area_light->pdf_li(prev_interaction_light_sample_ctx, ray.d);
            FloatType weight_light = power_heuristic(1, pdf_bsdf, 1, pdf_light);

            path_state->L[path_idx] += beta * weight_light * Le;
        }
    }

    // for active paths: advance one segment

    path_state->path_length[path_idx] += 1;

    switch (isect.material->get_material_type()) {

    case Material::Type::conductor: {
        queues->conductor_material->append_path(path_idx);
        return;
    }

    case Material::Type::coated_conductor: {
        queues->coated_conductor_material->append_path(path_idx);
        return;
    }

    case Material::Type::coated_diffuse: {
        queues->coated_diffuse_material->append_path(path_idx);
        return;
    }

    case Material::Type::dielectric: {
        queues->dielectric_material->append_path(path_idx);
        return;
    }

    case Material::Type::diffuse: {
        queues->diffuse_material->append_path(path_idx);
        return;
    }

    case Material::Type::diffuse_transmission: {
        queues->diffuse_transmission_material->append_path(path_idx);
        return;
    }

    case Material::Type::mix: {
        printf("\nyou should not see MixMaterial here\n\n");
        REPORT_FATAL_ERROR();
    }
    }

    REPORT_FATAL_ERROR();
}

__global__ void write_frame_buffer(Film *film, WavefrontPathIntegrator::Queues *queues) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->frame_buffer_counter) {
        return;
    }

    const auto pixel_idx = queues->frame_buffer_queue[queue_idx].pixel_idx;
    if (queue_idx > 0 && pixel_idx == queues->frame_buffer_queue[queue_idx - 1].pixel_idx) {
        return;
    }

    for (uint idx = queue_idx; idx < queues->frame_buffer_counter &&
                               queues->frame_buffer_queue[idx].pixel_idx == pixel_idx;
         ++idx) {
        // make sure the same pixels are written by the same thread
        const auto &frame_buffer = queues->frame_buffer_queue[idx];
        film->add_sample(frame_buffer.pixel_idx, frame_buffer.radiance, frame_buffer.lambda,
                         frame_buffer.weight);
    }
}

__global__ void fill_new_path_queue(WavefrontPathIntegrator::Queues *queues) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }

    queues->new_paths->queue_array[worker_idx] = worker_idx;
}

__global__ void generate_new_path(WavefrontPathIntegrator::PathState *path_state,
                                  WavefrontPathIntegrator::Queues *queues,
                                  const IntegratorBase *base) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->new_paths->counter) {
        return;
    }

    const uint path_idx = queues->new_paths->queue_array[queue_idx];

    const auto unique_path_id = atomicAdd(&path_state->global_path_counter, 1);
    if (unique_path_id >= path_state->total_path_num) {
        path_state->finished[path_idx] = true;
        return;
    }

    const uint width = path_state->image_resolution.x;
    const uint height = path_state->image_resolution.y;

    const uint pixel_idx = unique_path_id % (width * height);
    const uint sample_idx = unique_path_id / (width * height);

    auto sampler = &path_state->samplers[path_idx];

    auto p_pixel = Point2i(pixel_idx % width, pixel_idx / width);

    sampler->start_pixel_sample(pixel_idx, sample_idx, 0);

    path_state->camera_samples[path_idx] = sampler->get_camera_sample(p_pixel, base->filter);
    auto lu = sampler->get_1d();
    path_state->lambdas[path_idx] = SampledWavelengths::sample_visible(lu);

    path_state->camera_rays[path_idx] =
        base->camera->generate_ray(path_state->camera_samples[path_idx], sampler);

    path_state->pixel_indices[path_idx] = pixel_idx;
    path_state->sample_indices[path_idx] = sample_idx;
    path_state->path_length[path_idx] = 0;

    path_state->init_new_path(path_idx);

    queues->rays->append_path(path_idx);
}

__global__ void gpu_evaluate_material(WavefrontPathIntegrator::Queues::SingleQueue *material_queue,
                                      WavefrontPathIntegrator *integrator) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= material_queue->counter) {
        return;
    }

    auto path_state = &integrator->path_state;

    const uint path_idx = material_queue->queue_array[queue_idx];

    auto &lambda = path_state->lambdas[path_idx];

    auto sampler = &path_state->samplers[path_idx];

    auto &isect = path_state->shape_intersections[path_idx]->interaction;

    path_state->bsdf[path_idx] =
        isect.get_bsdf(lambda, integrator->base->camera, sampler->get_samples_per_pixel());

    integrator->sample_bsdf(path_idx, path_state);

    integrator->queues.rays->append_path(path_idx);
}

__global__ void ray_cast(WavefrontPathIntegrator::PathState *path_state,
                         WavefrontPathIntegrator::Queues *queues, const IntegratorBase *base) {
    const uint ray_queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_queue_idx >= queues->rays->counter) {
        return;
    }

    const uint path_idx = queues->rays->queue_array[ray_queue_idx];

    const auto camera_ray = path_state->camera_rays[path_idx];

    path_state->shape_intersections[path_idx] = base->intersect(camera_ray.ray, Infinity);
}

PBRT_CPU_GPU
void WavefrontPathIntegrator::sample_bsdf(uint path_idx, PathState *path_state) const {
    auto &isect = path_state->shape_intersections[path_idx]->interaction;
    auto &lambda = path_state->lambdas[path_idx];

    auto &ray = path_state->camera_rays[path_idx].ray;
    auto sampler = &path_state->samplers[path_idx];

    if (regularize && path_state->mis_parameters[path_idx].any_non_specular_bounces) {
        path_state->bsdf[path_idx].regularize();
    }

    if (pbrt::is_non_specular(path_state->bsdf[path_idx].flags())) {
        SampledSpectrum Ld = sample_ld(isect, &path_state->bsdf[path_idx], lambda, sampler);
        path_state->L[path_idx] += path_state->beta[path_idx] * Ld;
    }

    // Sample BSDF to get new path direction
    Vector3f wo = -ray.d;
    FloatType u = sampler->get_1d();
    auto bs = path_state->bsdf[path_idx].sample_f(wo, u, sampler->get_2d());
    if (!bs) {
        path_state->beta[path_idx] = SampledSpectrum(0.0);
        return;
    }

    path_state->beta[path_idx] *= bs->f * bs->wi.abs_dot(isect.shading.n.to_vector3()) / bs->pdf;

    path_state->mis_parameters[path_idx].pdf_bsdf =
        bs->pdf_is_proportional ? path_state->bsdf[path_idx].pdf(wo, bs->wi) : bs->pdf;
    path_state->mis_parameters[path_idx].specular_bounce = bs->is_specular();
    path_state->mis_parameters[path_idx].any_non_specular_bounces |= (!bs->is_specular());

    if (bs->is_transmission()) {
        path_state->mis_parameters[path_idx].eta_scale *= sqr(bs->eta);
    }

    path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx = isect;

    path_state->camera_rays[path_idx].ray = isect.spawn_ray(bs->wi);
}

void WavefrontPathIntegrator::evaluate_material(const Material::Type material_type) {
    auto material_queue = this->queues.get_material_queue(material_type);
    if (material_queue->counter <= 0) {
        return;
    }

    constexpr uint threads = 256;
    const auto blocks = divide_and_ceil(material_queue->counter, threads);

    gpu_evaluate_material<<<blocks, threads>>>(material_queue, this);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

PBRT_CPU_GPU
void WavefrontPathIntegrator::PathState::init_new_path(uint path_idx) {
    finished[path_idx] = false;
    shape_intersections[path_idx].reset();

    L[path_idx] = SampledSpectrum(0.0);
    beta[path_idx] = SampledSpectrum(1.0);
    path_length[path_idx] = 0;

    mis_parameters[path_idx].init();
}

void WavefrontPathIntegrator::PathState::create(uint samples_per_pixel, const Point2i &_resolution,
                                                const std::string &sampler_type,
                                                GPUMemoryAllocator &allocator) {
    image_resolution = _resolution;
    global_path_counter = 0;
    total_path_num = samples_per_pixel * image_resolution.x * image_resolution.y;

    camera_samples = allocator.allocate<CameraSample>(PATH_POOL_SIZE);
    camera_rays = allocator.allocate<CameraRay>(PATH_POOL_SIZE);
    lambdas = allocator.allocate<SampledWavelengths>(PATH_POOL_SIZE);

    L = allocator.allocate<SampledSpectrum>(PATH_POOL_SIZE);
    beta = allocator.allocate<SampledSpectrum>(PATH_POOL_SIZE);
    shape_intersections = allocator.allocate<pbrt::optional<ShapeIntersection>>(PATH_POOL_SIZE);

    path_length = allocator.allocate<uint>(PATH_POOL_SIZE);
    finished = allocator.allocate<bool>(PATH_POOL_SIZE);
    pixel_indices = allocator.allocate<uint>(PATH_POOL_SIZE);
    sample_indices = allocator.allocate<uint>(PATH_POOL_SIZE);

    bsdf = allocator.allocate<BSDF>(PATH_POOL_SIZE);
    mis_parameters = allocator.allocate<MISParameter>(PATH_POOL_SIZE);
    samplers = allocator.allocate<Sampler>(PATH_POOL_SIZE);

    constexpr uint threads = 1024;
    uint blocks = divide_and_ceil<uint>(PATH_POOL_SIZE, threads);

    if (sampler_type == "stratified") {
        const auto samples_per_dimension = static_cast<int>(std::sqrt(samples_per_pixel));
        if (samples_per_dimension * samples_per_dimension != samples_per_pixel) {
            REPORT_FATAL_ERROR();
        }

        auto stratified_samplers = allocator.allocate<StratifiedSampler>(PATH_POOL_SIZE);

        gpu_init_stratified_samplers<<<blocks, threads>>>(samplers, stratified_samplers,
                                                          samples_per_dimension, PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else if (sampler_type == "independent") {
        auto independent_samplers = allocator.allocate<IndependentSampler>(PATH_POOL_SIZE);

        gpu_init_independent_samplers<<<blocks, threads>>>(samplers, independent_samplers,
                                                           PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else {
        REPORT_FATAL_ERROR();
    }

    gpu_init_path_state<<<PATH_POOL_SIZE, threads>>>(this);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

void WavefrontPathIntegrator::Queues::init(GPUMemoryAllocator &allocator) {
    new_paths = build_new_queue(allocator);
    rays = build_new_queue(allocator);
    conductor_material = build_new_queue(allocator);
    coated_conductor_material = build_new_queue(allocator);
    coated_diffuse_material = build_new_queue(allocator);
    dielectric_material = build_new_queue(allocator);
    diffuse_material = build_new_queue(allocator);
    diffuse_transmission_material = build_new_queue(allocator);

    frame_buffer_counter = 0;
    frame_buffer_queue = allocator.allocate<FrameBuffer>(PATH_POOL_SIZE);
}

WavefrontPathIntegrator::Queues::SingleQueue *
WavefrontPathIntegrator::Queues::build_new_queue(GPUMemoryAllocator &allocator) {
    auto queue = allocator.allocate<SingleQueue>(PATH_POOL_SIZE);
    queue->counter = 0;
    queue->queue_array = allocator.allocate<uint>(PATH_POOL_SIZE);

    return queue;
}

WavefrontPathIntegrator *WavefrontPathIntegrator::create(uint samples_per_pixel,
                                                         const std::string &sampler_type,
                                                         const ParameterDictionary &parameters,
                                                         const IntegratorBase *base,
                                                         GPUMemoryAllocator &allocator) {
    auto integrator = allocator.allocate<WavefrontPathIntegrator>();

    integrator->samples_per_pixel = samples_per_pixel;

    integrator->base = base;
    integrator->path_state.create(samples_per_pixel, base->camera->get_camerabase()->resolution,
                                  sampler_type, allocator);

    integrator->queues.init(allocator);

    integrator->max_depth = parameters.get_integer("maxdepth", 5);
    integrator->regularize = parameters.get_bool("regularize", false);

    return integrator;
}

PBRT_CPU_GPU
SampledSpectrum WavefrontPathIntegrator::sample_ld(const SurfaceInteraction &intr, const BSDF *bsdf,
                                                   SampledWavelengths &lambda,
                                                   Sampler *sampler) const {
    // Initialize _LightSampleContext_ for light sampling
    LightSampleContext ctx(intr);
    // Try to nudge the light sampling position to correct side of the surface
    BxDFFlags flags = bsdf->flags();
    if (pbrt::is_reflective(flags) && !pbrt::is_transmissive(flags)) {
        ctx.pi = intr.offset_ray_origin(intr.wo);
    } else if (pbrt::is_transmissive(flags) && !pbrt::is_reflective(flags)) {
        ctx.pi = intr.offset_ray_origin(-intr.wo);
    }

    // Choose a light source for the direct lighting calculation
    FloatType u = sampler->get_1d();
    auto sampled_light = base->light_sampler->sample(ctx, u);

    Point2f uLight = sampler->get_2d();
    if (!sampled_light) {
        return SampledSpectrum(0);
    }

    // Sample a point on the light source for direct lighting
    auto light = sampled_light->light;
    auto ls = light->sample_li(ctx, uLight, lambda);
    if (!ls || !ls->l.is_positive() || ls->pdf == 0) {
        return SampledSpectrum(0);
    }

    // Evaluate BSDF for light sample and check light visibility
    Vector3f wo = intr.wo;
    Vector3f wi = ls->wi;
    SampledSpectrum f = bsdf->f(wo, wi) * wi.abs_dot(intr.shading.n.to_vector3());

    if (!f.is_positive() || !base->unoccluded(intr, ls->p_light)) {
        return SampledSpectrum(0);
    }

    // Return light's contribution to reflected radiance
    FloatType pdf_light = sampled_light->p * ls->pdf;
    if (pbrt::is_delta_light(light->get_light_type())) {
        return ls->l * f / pdf_light;
    }

    // for non delta light
    FloatType pdf_bsdf = bsdf->pdf(wo, wi);
    FloatType weight_light = power_heuristic(1, pdf_light, 1, pdf_bsdf);

    return weight_light * ls->l * f / pdf_light;
}

void WavefrontPathIntegrator::render(Film *film, const bool preview) {
    printf("wavefront: path pool size: %u\n", PATH_POOL_SIZE);

    const auto image_resolution = this->path_state.image_resolution;

    const auto num_pixels = image_resolution.x * image_resolution.y;

    GLHelper gl_helper;
    if (preview) {
        gl_helper.init("initializing", image_resolution);
    }

    constexpr uint threads = 256;

    // generate new paths for the whole pool
    fill_new_path_queue<<<PATH_POOL_SIZE, threads>>>(&queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    queues.new_paths->counter = PATH_POOL_SIZE;
    queues.rays->counter = 0;

    generate_new_path<<<divide_and_ceil(queues.new_paths->counter, threads), threads>>>(
        &path_state, &queues, base);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    while (queues.rays->counter > 0) {
        ray_cast<<<divide_and_ceil(queues.rays->counter, threads), threads>>>(&path_state, &queues,
                                                                              base);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // clear all queues before control stage
        for (auto _queue : queues.get_all_queues()) {
            _queue->counter = 0;
        }
        queues.frame_buffer_counter = 0;

        control_logic<<<divide_and_ceil(PATH_POOL_SIZE, threads), threads>>>(&path_state, &queues,
                                                                             max_depth, base);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        if (queues.frame_buffer_counter > 0) {
            // sort to make film writing deterministic
            std::sort(queues.frame_buffer_queue + 0,
                      queues.frame_buffer_queue + queues.frame_buffer_counter, std::less{});

            write_frame_buffer<<<divide_and_ceil(queues.frame_buffer_counter, threads), threads>>>(
                film, &queues);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());

            if (preview) {
                film->copy_to_frame_buffer(gl_helper.gpu_frame_buffer);

                const auto current_sample_idx =
                    std::min<uint>(path_state.global_path_counter / num_pixels, samples_per_pixel);

                gl_helper.draw_frame(
                    GLHelper::assemble_title(FloatType(current_sample_idx) / samples_per_pixel));
            }
        }

        if (queues.new_paths->counter > 0) {
            generate_new_path<<<divide_and_ceil(queues.new_paths->counter, threads), threads>>>(
                &path_state, &queues, base);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }

        for (const auto material_type : Material::get_all_material_type()) {
            evaluate_material(material_type);
        }
    }
}
