#include "hip/hip_runtime.h"
#include <pbrt/base/interaction.h>
#include <pbrt/base/shape.h>
#include <pbrt/shapes/triangle.h>

PBRT_CPU_GPU
pbrt::optional<ShapeIntersection> Triangle::intersect(const Ray &ray, FloatType t_max) const {
    Point3f points[3];
    get_points(points);

    auto tri_intersection = intersect_triangle(ray, t_max, points[0], points[1], points[2]);
    if (!tri_intersection) {
        return {};
    }

    SurfaceInteraction si = interaction_from_intersection(tri_intersection.value(), -ray.d);

    return ShapeIntersection(si, tri_intersection->t);
}

PBRT_CPU_GPU
FloatType Triangle::pdf(const Interaction &in) const {
    return 1.0 / this->area();
}

PBRT_CPU_GPU
FloatType Triangle::pdf(const ShapeSampleContext &ctx, const Vector3f &wi) const {
    FloatType solidAngle = solid_angle(ctx.p());
    // Return PDF based on uniform area sampling for challenging triangles
    if (solidAngle < MinSphericalSampleArea || solidAngle > MaxSphericalSampleArea) {
        // Intersect sample ray with shape geometry
        Ray ray = ctx.spawn_ray(wi);

        auto isect = intersect(ray, Infinity);
        if (!isect) {
            return 0;
        }

        // Compute PDF in solid angle measure from shape intersection point
        FloatType pdf = (1 / area()) / (isect->interaction.n.abs_dot(-wi) /
                                        ctx.p().squared_distance(isect->interaction.p()));

        if (isinf(pdf)) {
            pdf = 0;
        }

        return pdf;
    }

    FloatType pdf = 1 / solidAngle;
    // Adjust PDF for warp product sampling of triangle $\cos\theta$ factor
    if (ctx.ns != Normal3f(0, 0, 0)) {
        // Get triangle vertices in _p0_, _p1_, and _p2_
        /*
        const TriangleMesh *mesh = GetMesh();
        const int *v = &mesh->vertexIndices[3 * triIndex];


        Point3f p0 = mesh->p[v[0]], p1 = mesh->p[v[1]], p2 = mesh->p[v[2]];
        */
        Point3f points[3];
        get_points(points);
        const auto p0 = points[0];
        const auto p1 = points[1];
        const auto p2 = points[2];

        Point2f u = InvertSphericalTriangleSample(points, ctx.p(), wi);

        // Compute $\cos\theta$-based weights _w_ at sample domain corners
        Point3f rp = ctx.p();
        Vector3f wi[3] = {(p0 - rp).normalize(), (p1 - rp).normalize(), (p2 - rp).normalize()};

        FloatType w[4] = {std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[1])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[1])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[0])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[2]))};

        pdf *= bilinear_pdf(u, w);
    }

    return pdf;
}

PBRT_CPU_GPU
pbrt::optional<ShapeSample> Triangle::sample(Point2f u) const {
    const int *v = &(mesh->vertex_indices[3 * triangle_idx]);
    const Point3f p0 = mesh->p[v[0]];
    const Point3f p1 = mesh->p[v[1]];
    const Point3f p2 = mesh->p[v[2]];

    FloatType b[3];
    sample_uniform_triangle(b, u);
    const Point3f p = b[0] * p0 + b[1] * p1 + b[2] * p2;

    // Compute surface normal for sampled point on triangle
    // Normal3f n = Normalize(Normal3f(Cross(p1 - p0, p2 - p0)));
    Normal3f n = Normal3f((p1 - p0).cross(p2 - p0).normalize());

    if (mesh->n) {
        Normal3f ns =
            (b[0] * mesh->n[v[0]] + b[1] * mesh->n[v[1]] + (1 - b[0] - b[1]) * mesh->n[v[2]]);
        n = n.face_forward(ns);
    } else if ((mesh->reverse_orientation ^ mesh->transformSwapsHandedness)) {
        // this part not implemented
        n *= -1;
    }

    Point2f uv[3];
    if (mesh->uv) {
        for (uint idx = 0; idx < 3; ++idx) {
            uv[idx] = mesh->uv[v[idx]];
        }
    } else {
        uv[0] = Point2f(0, 0);
        uv[1] = Point2f(1, 0);
        uv[2] = Point2f(1, 1);
    }

    Point2f uvSample = b[0] * uv[0] + b[1] * uv[1] + b[2] * uv[2];

    // Compute error bounds _pError_ for sampled point on triangle
    Point3f pAbsSum = (b[0] * p0).abs() + (b[1] * p1).abs() + ((1 - b[0] - b[1]) * p2).abs();

    Vector3f pError = (gamma(6) * pAbsSum).to_vector3();

    return ShapeSample{
        .interaction = Interaction(Point3fi(p, pError), n, uvSample),
        .pdf = FloatType(1.0) / area(),
    };
}

PBRT_CPU_GPU
pbrt::optional<ShapeSample> Triangle::sample(const ShapeSampleContext &ctx, Point2f u) const {
    Point3f points[3];
    get_points(points);
    const auto p0 = points[0];
    const auto p1 = points[1];
    const auto p2 = points[2];
    const int *v = &(mesh->vertex_indices[3 * triangle_idx]);

    // Use uniform area sampling for numerically unstable cases
    FloatType solid_angle = this->solid_angle(ctx.p());

    if (solid_angle < MinSphericalSampleArea || solid_angle > MaxSphericalSampleArea) {
        auto ss = sample(u);
        Vector3f wi = ss->interaction.p() - ctx.p();
        if (wi.squared_length() == 0) {
            return {};
        }
        wi = wi.normalize();

        // Convert area sampling PDF in _ss_ to solid angle measure
        ss->pdf /=
            ss->interaction.n.abs_dot(-wi) / (ctx.p() - ss->interaction.p()).squared_length();

        if (is_inf(ss->pdf)) {
            return {};
        }

        return ss;
    }

    // Sample spherical triangle from reference point
    // Apply warp product sampling for cosine factor at reference point
    FloatType pdf = 1.0;
    if (ctx.ns != Normal3f(0, 0, 0)) {
        // Compute $\cos\theta$-based weights _w_ at sample domain corners
        Point3f rp = ctx.p();
        Vector3f wi[3] = {(p0 - rp).normalize(), (p1 - rp).normalize(), (p2 - rp).normalize()};

        FloatType w[4] = {std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[1])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[1])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[0])),
                          std::max<FloatType>(0.01, ctx.ns.abs_dot(wi[2]))};
        u = sample_bilinear(u, w);
        pdf = bilinear_pdf(u, w);
    }

    FloatType triPDF;
    FloatType b[3];
    sample_spherical_triangle(b, &triPDF, points, ctx.p(), u);
    if (triPDF == 0.0) {
        return {};
    }
    pdf *= triPDF;

    // Compute error bounds _pError_ for sampled point on triangle
    Point3f pAbsSum = (b[0] * p0).abs() + (b[1] * p1).abs() + ((1 - b[0] - b[1]) * p2).abs();
    Vector3f pError = (gamma(6) * pAbsSum).to_vector3();

    // Return _ShapeSample_ for solid angle sampled point on triangle
    Point3f p = b[0] * p0 + b[1] * p1 + b[2] * p2;
    // Compute surface normal for sampled point on triangle
    Normal3f n = Normal3f((p1 - p0).cross(p2 - p0).normalize());

    if (mesh->n) {
        Normal3f ns(b[0] * mesh->n[v[0]] + b[1] * mesh->n[v[1]] +
                    (1 - b[0] - b[1]) * mesh->n[v[2]]);
        n = n.face_forward(ns);
    } else if (mesh->reverse_orientation ^ mesh->transformSwapsHandedness) {
        n *= -1;
    }

    Point2f uv[3];
    if (mesh->uv) {
        uv[0] = mesh->uv[v[0]];
        uv[1] = mesh->uv[v[1]];
        uv[2] = mesh->uv[v[2]];
    } else {
        uv[0] = Point2f(0, 0);
        uv[1] = Point2f(1, 0);
        uv[2] = Point2f(1, 1);
    }

    Point2f uvSample = b[0] * uv[0] + b[1] * uv[1] + b[2] * uv[2];

    return ShapeSample{
        .interaction = Interaction(Point3fi(p, pError), n, uvSample),
        .pdf = pdf,
    };
}

PBRT_CPU_GPU
pbrt::optional<Triangle::TriangleIntersection>
Triangle::intersect_triangle(const Ray &ray, FloatType t_max, const Point3f &p0, const Point3f &p1,
                             const Point3f &p2) const {
    // Return no intersection if triangle is degenerate
    if ((p2 - p0).cross(p1 - p0).squared_length() == 0.0) {
        return {};
    }

    // Transform triangle vertices to ray coordinate space
    // Translate vertices based on ray origin
    Point3f p0t = p0 - ray.o.to_vector3();
    Point3f p1t = p1 - ray.o.to_vector3();
    Point3f p2t = p2 - ray.o.to_vector3();

    // Permute components of triangle vertices and ray direction
    uint8_t kz = ray.d.abs().max_component_index();
    uint8_t kx = (kz + 1) % 3;
    uint8_t ky = (kz + 2) % 3;

    uint8_t permuted_idx[3] = {kx, ky, kz};
    Vector3f d = ray.d.permute(permuted_idx);

    p0t = p0t.permute(permuted_idx);
    p1t = p1t.permute(permuted_idx);
    p2t = p2t.permute(permuted_idx);

    // Apply shear transformation to translated vertex positions
    FloatType Sx = -d.x / d.z;
    FloatType Sy = -d.y / d.z;
    FloatType Sz = 1 / d.z;
    p0t.x += Sx * p0t.z;
    p0t.y += Sy * p0t.z;
    p1t.x += Sx * p1t.z;
    p1t.y += Sy * p1t.z;
    p2t.x += Sx * p2t.z;
    p2t.y += Sy * p2t.z;

    // Compute edge function coefficients _e0_, _e1_, and _e2_
    FloatType e0 = difference_of_products(p1t.x, p2t.y, p1t.y, p2t.x);
    FloatType e1 = difference_of_products(p2t.x, p0t.y, p2t.y, p0t.x);
    FloatType e2 = difference_of_products(p0t.x, p1t.y, p0t.y, p1t.x);

    // Fall back to double-precision test at triangle edges
    if (sizeof(FloatType) == sizeof(float) && (e0 == 0.0f || e1 == 0.0f || e2 == 0.0f)) {
        double p2txp1ty = (double)p2t.x * (double)p1t.y;
        double p2typ1tx = (double)p2t.y * (double)p1t.x;
        e0 = (FloatType)(p2typ1tx - p2txp1ty);
        double p0txp2ty = (double)p0t.x * (double)p2t.y;
        double p0typ2tx = (double)p0t.y * (double)p2t.x;
        e1 = (FloatType)(p0typ2tx - p0txp2ty);
        double p1txp0ty = (double)p1t.x * (double)p0t.y;
        double p1typ0tx = (double)p1t.y * (double)p0t.x;
        e2 = (FloatType)(p1typ0tx - p1txp0ty);
    }

    // Perform triangle edge and determinant tests
    if ((e0 < 0 || e1 < 0 || e2 < 0) && (e0 > 0 || e1 > 0 || e2 > 0)) {
        return {};
    }

    FloatType det = e0 + e1 + e2;
    if (det == 0) {
        return {};
    }

    // Compute scaled hit distance to triangle and test against ray $t$ range
    p0t.z *= Sz;
    p1t.z *= Sz;
    p2t.z *= Sz;
    FloatType tScaled = e0 * p0t.z + e1 * p1t.z + e2 * p2t.z;
    if (det < 0 && (tScaled >= 0 || tScaled < t_max * det)) {
        return {};
    }

    if (det > 0 && (tScaled <= 0 || tScaled > t_max * det)) {
        return {};
    }

    // Compute barycentric coordinates and $t$ value for triangle intersection
    FloatType invDet = 1 / det;
    FloatType b0 = e0 * invDet, b1 = e1 * invDet, b2 = e2 * invDet;
    FloatType t = tScaled * invDet;

    // Ensure that computed triangle $t$ is conservatively greater than zero
    // Compute $\delta_z$ term for triangle $t$ error bounds
    // FloatType maxZt = MaxComponentValue(Abs(Vector3f(p0t.z, p1t.z, p2t.z)));

    FloatType maxZt = Vector3f(p0t.z, p1t.z, p2t.z).abs().max_component_value();
    FloatType deltaZ = gamma(3) * maxZt;

    // Compute $\delta_x$ and $\delta_y$ terms for triangle $t$ error bounds
    FloatType maxXt = Vector3f(p0t.x, p1t.x, p2t.x).abs().max_component_value();
    FloatType maxYt = Vector3f(p0t.y, p1t.y, p2t.y).abs().max_component_value();

    FloatType deltaX = gamma(5) * (maxXt + maxZt);
    FloatType deltaY = gamma(5) * (maxYt + maxZt);

    // Compute $\delta_e$ term for triangle $t$ error bounds
    FloatType deltaE = 2 * (gamma(2) * maxXt * maxYt + deltaY * maxXt + deltaX * maxYt);

    // Compute $\delta_t$ term for triangle $t$ error bounds and check _t_
    FloatType maxE = Vector3f(e0, e1, e2).abs().max_component_value();
    FloatType deltaT =
        3 * (gamma(3) * maxE * maxZt + deltaE * maxZt + deltaZ * maxE) * std::abs(invDet);
    if (t <= deltaT) {
        return {};
    }

    // Return _TriangleIntersection_ for intersection
    return TriangleIntersection(b0, b1, b2, t);
}

PBRT_CPU_GPU
SurfaceInteraction Triangle::interaction_from_intersection(const TriangleIntersection &ti,
                                                           const Vector3f &wo) const {
    const int *v = &(mesh->vertex_indices[3 * triangle_idx]);
    const Point3f p0 = mesh->p[v[0]];
    const Point3f p1 = mesh->p[v[1]];
    const Point3f p2 = mesh->p[v[2]];

    // Compute triangle partial derivatives
    // Compute deltas and matrix determinant for triangle partial derivatives
    // Get triangle texture coordinates in _uv_ array

    Point2f uv[3];
    if (mesh->uv) {
        for (uint idx = 0; idx < 3; ++idx) {
            uv[idx] = mesh->uv[v[idx]];
        }
    } else {
        uv[0] = Point2f(0, 0);
        uv[1] = Point2f(1, 0);
        uv[2] = Point2f(1, 1);
    }

    Vector2f duv02 = uv[0] - uv[2], duv12 = uv[1] - uv[2];
    Vector3f dp02 = p0 - p2;
    Vector3f dp12 = p1 - p2;
    FloatType determinant = difference_of_products(duv02[0], duv12[1], duv02[1], duv12[0]);

    Vector3f dpdu, dpdv;
    bool degenerateUV = std::abs(determinant) < 1e-9f;
    if (!degenerateUV) {
        // Compute triangle $\dpdu$ and $\dpdv$ via matrix inversion
        FloatType invdet = 1 / determinant;
        dpdu = difference_of_products(duv12[1], dp02, duv02[1], dp12) * invdet;
        dpdv = difference_of_products(duv02[0], dp12, duv12[0], dp02) * invdet;
    }
    // Handle degenerate triangle $(u,v)$ parameterization or partial derivatives
    if (degenerateUV || dpdu.cross(dpdv).squared_length() == 0) {
        Vector3f ng = (p2 - p0).cross(p1 - p0);
        if (ng.squared_length() == 0) {
            ng = (p2 - p0).cross(p1 - p0);
        }
        ng.normalize().coordinate_system(&dpdu, &dpdv);
    }

    // Interpolate $(u,v)$ parametric coordinates and hit point
    Point3f pHit = ti.b0 * p0 + ti.b1 * p1 + ti.b2 * p2;
    Point2f uvHit = ti.b0 * uv[0] + ti.b1 * uv[1] + ti.b2 * uv[2];

    bool flipNormal = mesh->reverse_orientation ^ mesh->transformSwapsHandedness;
    // Compute error bounds _pError_ for triangle intersection
    Point3f pAbsSum = (ti.b0 * p0).abs() + (ti.b1 * p1).abs() + (ti.b2 * p2).abs();
    Vector3f pError = gamma(7) * pAbsSum.to_vector3();

    SurfaceInteraction isect(Point3fi(pHit, pError), uvHit, wo, dpdu, dpdv, Normal3f(), Normal3f(),
                             flipNormal);

    isect.faceIndex = mesh->faceIndices ? mesh->faceIndices[triangle_idx] : 0;

    // Set final surface normal and shading geometry for triangle
    // Override surface normal in _isect_ for triangle
    isect.n = Normal3f(dp02.cross(dp12).normalize());
    isect.shading.n = isect.n;

    if (mesh->reverse_orientation ^ mesh->transformSwapsHandedness) {
        isect.n = isect.shading.n = -isect.n;
    }

    if (mesh->n || mesh->s) {
        // Initialize _Triangle_ shading geometry
        // Compute shading normal _ns_ for triangle
        Normal3f ns;
        if (mesh->n) {
            ns = ti.b0 * mesh->n[v[0]] + ti.b1 * mesh->n[v[1]] + ti.b2 * mesh->n[v[2]];
            ns = ns.squared_length() > 0 ? ns.normalize() : isect.n;
        } else {
            ns = isect.n;
        }

        // Compute shading tangent _ss_ for triangle
        Vector3f ss;
        if (mesh->s) {
            ss = ti.b0 * mesh->s[v[0]] + ti.b1 * mesh->s[v[1]] + ti.b2 * mesh->s[v[2]];
            if (ss.squared_length() == 0) {
                ss = isect.dpdu;
            }
        } else {
            ss = isect.dpdu;
        }

        // Compute shading bitangent _ts_ for triangle and adjust _ss_
        auto ts = ns.to_vector3().cross(ss);
        if (ts.squared_length() > 0) {
            ss = ts.cross(ns.to_vector3());
        } else {
            ns.to_vector3().coordinate_system(&ss, &ts);
        }

        // Compute $\dndu$ and $\dndv$ for triangle shading geometry
        Normal3f dndu, dndv;
        if (mesh->n) {
            // Compute deltas for triangle partial derivatives of normal
            Vector2f duv02 = uv[0] - uv[2];
            Vector2f duv12 = uv[1] - uv[2];
            Normal3f dn1 = mesh->n[v[0]] - mesh->n[v[2]];
            Normal3f dn2 = mesh->n[v[1]] - mesh->n[v[2]];

            auto determinant = difference_of_products(duv02[0], duv12[1], duv02[1], duv12[0]);
            bool degenerateUV = std::abs(determinant) < 1e-9;
            if (degenerateUV) {
                // We can still compute dndu and dndv, with respect to the
                // same arbitrary coordinate system we use to compute dpdu
                // and dpdv when this happens. It's important to do this
                // (rather than giving up) so that ray differentials for
                // rays reflected from triangles with degenerate
                // parameterizations are still reasonable.

                auto dn = (mesh->n[v[2]] - mesh->n[v[0]]).cross(mesh->n[v[1]] - mesh->n[v[0]]);

                if (dn.squared_length() == 0) {
                    dndu = dndv = Normal3f(0, 0, 0);
                } else {
                    Vector3f dnu, dnv;
                    dn.coordinate_system(&dnu, &dnv);
                    dndu = Normal3f(dnu);
                    dndv = Normal3f(dnv);
                }
            } else {
                auto invDet = 1 / determinant;
                dndu = difference_of_products(duv12[1], dn1, duv02[1], dn2) * invDet;
                dndv = difference_of_products(duv02[0], dn2, duv12[0], dn1) * invDet;
            }
        } else {
            dndu = dndv = Normal3f(0, 0, 0);
        }

        isect.set_shading_geometry(ns, ss, ts, dndu, dndv, true);
    }

    return isect;
}
