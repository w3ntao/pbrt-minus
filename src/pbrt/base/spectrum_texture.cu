#include <pbrt/base/spectrum_texture.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/spectrum_util/rgb_color_space.h>
#include <pbrt/textures/spectrum_constant_texture.h>
#include <pbrt/textures/spectrum_image_texture.h>
#include <pbrt/textures/spectrum_scaled_texture.h>

const SpectrumTexture *
SpectrumTexture::create(const std::string &texture_type, const SpectrumType spectrum_type,
                        const Transform &render_from_object, const RGBColorSpace *color_space,
                        const ParameterDictionary &parameters, GPUMemoryAllocator &allocator) {
    if (texture_type == "imagemap") {
        auto image_texture = SpectrumImageTexture::create(spectrum_type, render_from_object,
                                                          color_space, parameters, allocator);

        auto spectrum_texture = allocator.allocate<SpectrumTexture>();

        spectrum_texture->init(image_texture);
        return spectrum_texture;
    }

    if (texture_type == "scale") {
        auto scaled_texture = allocator.allocate<SpectrumScaledTexture>();
        auto spectrum_texture = allocator.allocate<SpectrumTexture>();

        scaled_texture->init(spectrum_type, parameters, allocator);
        spectrum_texture->init(scaled_texture);
        return spectrum_texture;
    }

    printf("\ntexture type `%s` not implemented for SpectrumTexture\n", texture_type.c_str());

    REPORT_FATAL_ERROR();
    return nullptr;
}

const SpectrumTexture *
SpectrumTexture::create_constant_float_val_texture(FloatType val, GPUMemoryAllocator &allocator) {
    auto spectrum_constant_texture = allocator.allocate<SpectrumConstantTexture>();
    auto spectrum_texture = allocator.allocate<SpectrumTexture>();

    spectrum_constant_texture->init(Spectrum::create_constant_spectrum(val, allocator));
    spectrum_texture->init(spectrum_constant_texture);

    return spectrum_texture;
}

const SpectrumTexture *SpectrumTexture::create_constant_texture(const Spectrum *spectrum,
                                                                GPUMemoryAllocator &allocator) {
    if (spectrum == nullptr) {
        REPORT_FATAL_ERROR();
    }
    auto spectrum_constant_texture = allocator.allocate<SpectrumConstantTexture>();
    auto spectrum_texture = allocator.allocate<SpectrumTexture>();

    spectrum_constant_texture->init(spectrum);
    spectrum_texture->init(spectrum_constant_texture);

    return spectrum_texture;
}

void SpectrumTexture::init(const SpectrumConstantTexture *constant_texture) {
    type = Type::constant;
    ptr = constant_texture;
}

void SpectrumTexture::init(const SpectrumImageTexture *image_texture) {
    type = Type::image;
    ptr = image_texture;
}

void SpectrumTexture::init(const SpectrumScaledTexture *scale_texture) {
    type = Type::scaled;
    ptr = scale_texture;
}

PBRT_CPU_GPU
SampledSpectrum SpectrumTexture::evaluate(const TextureEvalContext &ctx,
                                          const SampledWavelengths &lambda) const {
    switch (type) {
    case Type::constant: {
        return static_cast<const SpectrumConstantTexture *>(ptr)->evaluate(ctx, lambda);
    }
    case Type::image: {
        return static_cast<const SpectrumImageTexture *>(ptr)->evaluate(ctx, lambda);
    }
    case Type::scaled: {
        return static_cast<const SpectrumScaledTexture *>(ptr)->evaluate(ctx, lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}
