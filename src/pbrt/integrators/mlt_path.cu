#include "hip/hip_runtime.h"
#include <numeric>
#include <pbrt/base/camera.h>
#include <pbrt/base/film.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/sampler.h>
#include <pbrt/films/grey_scale_film.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/gui/gl_helper.h>
#include <pbrt/integrators/megakernel_path.h>
#include <pbrt/integrators/mlt_path.h>
#include <pbrt/samplers/mlt.h>
#include <pbrt/scene/parameter_dictionary.h>
#include <pbrt/spectrum_util/global_spectra.h>

constexpr size_t NUM_MLT_PATH_SAMPLERS = 64 * 1024;
// large number of samplers: large number of shallow markov chains
// small number of samplers: small number of deep markov chains

struct MLTSample {
    PathSample path_sample;
    FloatType sampling_density;
    FloatType weight;

    PBRT_CPU_GPU
    MLTSample(const Point2f p_film, const SampledSpectrum &_radiance,
              const SampledWavelengths &_lambda, const FloatType _weight,
              const FloatType _sampling_density)
        : path_sample(PathSample(p_film, _radiance, _lambda)), weight(_weight),
          sampling_density(_sampling_density) {}
};

__global__ void build_bootstrap_samples(const uint num_paths_per_worker, double *luminance_per_path,
                                        const MLTPathIntegrator *mlt_integrator) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= NUM_MLT_PATH_SAMPLERS) {
        return;
    }

    auto sampler = &mlt_integrator->samplers[worker_idx];
    auto mlt_sampler = &mlt_integrator->mlt_samplers[worker_idx];

    for (int idx = 0; idx < num_paths_per_worker; idx++) {
        const auto path_idx = worker_idx * num_paths_per_worker + idx;

        mlt_sampler->init(path_idx);
        mlt_sampler->start_stream(0);

        const auto path_sample = mlt_integrator->generate_path_sample(sampler);

        const auto illuminance =
            mlt_integrator->compute_luminance(path_sample.radiance, path_sample.lambda);

        luminance_per_path[path_idx] = illuminance;
    }
}

__global__ void prepare_initial_state(PathSample *path_samples,
                                      const MLTPathIntegrator *mlt_integrator) {
    // this implementation is different from PBRT-v4:
    // in terms of selecting the 0th path samples (after bootstrap), PBRT-v4 did an importance
    // sampling from bootstrap (so multiple sampler might choose the same paths) but pbrt-minus
    // initiated each sampler with different bootstrap

    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= NUM_MLT_PATH_SAMPLERS) {
        return;
    }

    auto sampler = &mlt_integrator->samplers[worker_idx];
    auto mlt_sampler = &mlt_integrator->mlt_samplers[worker_idx];

    mlt_sampler->init(worker_idx);
    mlt_sampler->start_iteration();
    mlt_sampler->start_stream(0);

    path_samples[worker_idx] = mlt_integrator->generate_path_sample(sampler);
}

__global__ void wavefront_render(MLTSample *mlt_samples, const uint num_mutations,
                                 PathSample *path_samples, const MLTPathIntegrator *mlt_integrator,
                                 RNG *rngs) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num_mutations) {
        return;
    }

    auto sampler = &mlt_integrator->samplers[worker_idx];
    auto mlt_sampler = &mlt_integrator->mlt_samplers[worker_idx];
    auto rng = &rngs[worker_idx];

    mlt_sampler->start_iteration();
    mlt_sampler->start_stream(0);

    const auto proposed_path = mlt_integrator->generate_path_sample(sampler);
    const auto current_path = &path_samples[worker_idx];

    const auto proposed_c =
        mlt_integrator->compute_luminance(proposed_path.radiance, proposed_path.lambda);
    const auto current_c =
        mlt_integrator->compute_luminance(current_path->radiance, current_path->lambda);

    const auto sample_idx = worker_idx * 2;

    for (auto offset = 0; offset < 2; ++offset) {
        mlt_samples[sample_idx + offset].weight = 0;
        mlt_samples[sample_idx + offset].sampling_density = 0;
    }

    FloatType accept_prob = NAN;
    if (current_c == 0 && proposed_c == 0) {
        accept_prob = 0.5;

    } else if (current_c == 0) {
        accept_prob = 1;
        mlt_samples[sample_idx] = MLTSample(proposed_path.p_film, proposed_path.radiance,
                                            proposed_path.lambda, 1.0 / proposed_c, 1);

    } else if (proposed_c == 0) {
        accept_prob = 0;
        mlt_samples[sample_idx] = MLTSample(current_path->p_film, current_path->radiance,
                                            current_path->lambda, 1.0 / current_c, 1);
    } else {
        accept_prob = std::min<FloatType>(1.0, proposed_c / current_c);

        const FloatType proposed_path_weight = accept_prob / proposed_c;
        const FloatType current_path_weight = (1 - accept_prob) / current_c;

        mlt_samples[sample_idx + 0] =
            MLTSample(current_path->p_film, current_path->radiance, current_path->lambda,
                      current_path_weight, 1 - accept_prob);
        mlt_samples[sample_idx + 1] =
            MLTSample(proposed_path.p_film, proposed_path.radiance, proposed_path.lambda,
                      proposed_path_weight, accept_prob);
    }

    if (rng->uniform<FloatType>() < accept_prob) {
        *current_path = proposed_path;
        mlt_sampler->accept();

    } else {
        mlt_sampler->reject();
    }
}

MLTPathIntegrator *MLTPathIntegrator::create(const int mutations_per_pixel,
                                             const ParameterDictionary &parameters,
                                             const IntegratorBase *base,
                                             GPUMemoryAllocator &allocator) {
    auto integrator = allocator.allocate<MLTPathIntegrator>();

    integrator->base = base;
    integrator->mlt_samplers = allocator.allocate<MLTSampler>(NUM_MLT_PATH_SAMPLERS);
    integrator->samplers = allocator.allocate<Sampler>(NUM_MLT_PATH_SAMPLERS);

    const auto large_step_probability = parameters.get_float("largestepprobability", 0.3);
    const auto sigma = parameters.get_float("sigma", 0.01);

    for (uint idx = 0; idx < NUM_MLT_PATH_SAMPLERS; ++idx) {
        integrator->mlt_samplers[idx].setup_config(mutations_per_pixel, sigma,
                                                   large_step_probability, 1);
        integrator->samplers[idx].init(&integrator->mlt_samplers[idx]);
    }

    integrator->film_dimension = base->camera->get_camerabase()->resolution;
    integrator->cie_y = parameters.global_spectra->cie_y;

    integrator->max_depth = parameters.get_integer("maxdepth", 5);
    integrator->regularize = parameters.get_bool("regularize", false);

    return integrator;
}

PBRT_CPU_GPU
PathSample MLTPathIntegrator::generate_path_sample(Sampler *sampler) const {
    const auto lu = sampler->get_1d();
    auto lambda = SampledWavelengths::sample_visible(lu);

    const auto u = sampler->get_2d();
    const auto p_film = Point2f(u.x * film_dimension.x, u.y * film_dimension.y);
    const auto camera_sample = CameraSample(p_film, 1);

    const auto ray = base->camera->generate_ray(camera_sample, sampler);

    const auto radiance = ray.weight * MegakernelPathIntegrator::evaluate_li(
                                           ray.ray, lambda, base, sampler, max_depth, regularize);

    return PathSample(p_film, radiance, lambda);
}

double MLTPathIntegrator::render(Film *film, GreyScaleFilm &heat_map,
                                 const uint mutations_per_pixel, const bool preview) {
    GPUMemoryAllocator local_allocator;

    const auto num_paths_per_worker =
        divide_and_ceil<int>(film_dimension.x * film_dimension.y, NUM_MLT_PATH_SAMPLERS);

    const auto num_bootstrap_paths = num_paths_per_worker * NUM_MLT_PATH_SAMPLERS;

    if (num_paths_per_worker <= 0) {
        REPORT_FATAL_ERROR();
    }

    auto path_samples = local_allocator.allocate<PathSample>(NUM_MLT_PATH_SAMPLERS);
    auto luminance_per_path = local_allocator.allocate<double>(num_bootstrap_paths);

    constexpr uint threads = 64;
    const uint blocks = divide_and_ceil<uint>(NUM_MLT_PATH_SAMPLERS, threads);
    build_bootstrap_samples<<<blocks, threads>>>(num_paths_per_worker, luminance_per_path, this);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    const double sum_luminance =
        std::accumulate(luminance_per_path + 0, luminance_per_path + num_bootstrap_paths, 0.0);

    const double brightness = sum_luminance / num_bootstrap_paths;

    const long long total_mutations =
        static_cast<long long>(mutations_per_pixel) * film_dimension.x * film_dimension.y;

    const auto total_pass = divide_and_ceil<long long>(total_mutations, NUM_MLT_PATH_SAMPLERS);

    printf("MLT-PATH:\n");
    printf("    number of bootstrap paths: %lu\n", num_bootstrap_paths);
    printf("    MLT samplers: %lu\n", NUM_MLT_PATH_SAMPLERS);
    printf("    mutations per samplers: %.1f\n", double(total_mutations) / NUM_MLT_PATH_SAMPLERS);
    printf("    brightness: %.6f\n", brightness);

    if (brightness <= 0.0) {
        REPORT_FATAL_ERROR();
    }

    auto mlt_samples = local_allocator.allocate<MLTSample>(2 * NUM_MLT_PATH_SAMPLERS);

    GLHelper gl_helper;
    if (preview) {
        gl_helper.init("initializing", film_dimension);
    }

    auto rngs = local_allocator.allocate<RNG>(NUM_MLT_PATH_SAMPLERS);

    for (uint idx = 0; idx < NUM_MLT_PATH_SAMPLERS; ++idx) {
        rngs[idx].set_sequence(idx + NUM_MLT_PATH_SAMPLERS);
    }

    prepare_initial_state<<<blocks, threads>>>(path_samples, this);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    long long accumulate_samples = 0; // this is for debugging and verification
    for (uint pass = 0; pass < total_pass; ++pass) {
        const uint num_mutations = pass == total_pass - 1
                                       ? total_mutations - (total_pass - 1) * NUM_MLT_PATH_SAMPLERS
                                       : NUM_MLT_PATH_SAMPLERS;

        wavefront_render<<<blocks, threads>>>(mlt_samples, num_mutations, path_samples, this, rngs);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        for (uint idx = 0; idx < num_mutations * 2; ++idx) {
            accumulate_samples += 1;
            const auto path_sample = &mlt_samples[idx].path_sample;
            const auto p_film = path_sample->p_film;
            const auto weight = mlt_samples[idx].weight;
            const auto sampling_density = mlt_samples[idx].sampling_density;

            if (weight > 0) {
                film->add_splat(p_film, path_sample->radiance * weight, path_sample->lambda);
            }

            auto p_discrete = (p_film + Vector2f(0.5, 0.5)).floor();
            p_discrete.x = clamp<int>(p_discrete.x, 0, film_dimension.x - 1);
            p_discrete.y = clamp<int>(p_discrete.y, 0, film_dimension.y - 1);

            heat_map.add_sample(p_discrete, sampling_density);
        }

        if (preview) {
            film->copy_to_frame_buffer(gl_helper.gpu_frame_buffer,
                                       brightness / mutations_per_pixel);
            gl_helper.draw_frame(GLHelper::assemble_title(FloatType(pass + 1) / total_pass));
        }
    }

    if (accumulate_samples != total_mutations * 2) {
        REPORT_FATAL_ERROR();
    }

    return brightness;
}
