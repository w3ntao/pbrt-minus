#include "hip/hip_runtime.h"
#include <pbrt/base/light.h>
#include <pbrt/lights/spot_light.h>
#include <pbrt/scene/parameter_dictionary.h>
#include <pbrt/spectrum_util/global_spectra.h>
#include <pbrt/spectrum_util/rgb_color_space.h>

#include <pbrt/gpu/gpu_memory_allocator.h>

SpotLight *SpotLight::create(const Transform &renderFromLight,
                             const ParameterDictionary &parameters, GPUMemoryAllocator &allocator) {
    auto I = parameters.get_spectrum("I", SpectrumType::Illuminant, allocator);
    if (I == nullptr) {
        I = parameters.global_spectra->rgb_color_space->illuminant;
    }

    auto sc = parameters.get_float("scale", 1.0);
    auto coneangle = parameters.get_float("coneangle", 30.0);
    auto conedelta = parameters.get_float("conedeltaangle", 5.0);

    // Compute spotlight rendering to light transformation
    Point3f from = parameters.get_point3("from", Point3f(0, 0, 0));
    Point3f to = parameters.get_point3("to", Point3f(0, 0, 1));

    auto dirToZ = Transform(Frame::from_z((to - from).normalize()));
    auto t = Transform::translate(from.x, from.y, from.z) * dirToZ.inverse();
    auto finalRenderFromLight = renderFromLight * t;

    sc /= I->to_photometric(parameters.global_spectra->cie_y);
    auto phi_v = parameters.get_float("power", -1);
    if (phi_v > 0) {
        auto cosFalloffEnd = std::cos(degree_to_radian(coneangle));
        auto cosFalloffStart = std::cos(degree_to_radian(coneangle - conedelta));
        auto k_e =
            2 * compute_pi() * ((1 - cosFalloffStart) + (cosFalloffStart - cosFalloffEnd) / 2);
        sc *= phi_v / k_e;
    }

    auto spot_light = allocator.allocate<SpotLight>();

    spot_light->init(finalRenderFromLight, I, sc, coneangle, coneangle - conedelta);

    return spot_light;
}

void SpotLight::init(const Transform &renderFromLight, const Spectrum *Iemit, FloatType _scale,
                     FloatType totalWidth, FloatType falloffStart) {
    this->light_type = LightType::delta_position;
    this->render_from_light = renderFromLight;

    this->i_emit = Iemit;
    this->scale = _scale;

    this->cosFalloffEnd = std::cos(degree_to_radian(totalWidth));
    this->cosFalloffStart = std::cos(degree_to_radian(falloffStart));
}

PBRT_CPU_GPU
SampledSpectrum SpotLight::l(Point3f p, Normal3f n, Point2f uv, Vector3f w,
                             const SampledWavelengths &lambda) const {
    REPORT_FATAL_ERROR();
    return {};
}

PBRT_CPU_GPU
pbrt::optional<LightLiSample> SpotLight::sample_li(const LightSampleContext &ctx, const Point2f &u,
                                                   SampledWavelengths &lambda) const {
    Point3f p = render_from_light(Point3f(0, 0, 0));
    Vector3f wi = (p - ctx.p()).normalize();
    // Compute incident radiance _Li_ for _SpotLight_

    Vector3f wLight = (render_from_light.apply_inverse(-wi)).normalize();

    SampledSpectrum Li = I(wLight, lambda) / (p - ctx.p()).squared_length();

    if (!Li.is_positive()) {
        return {};
    }

    return LightLiSample(Li, wi, 1, Interaction(p));
}

PBRT_CPU_GPU
pbrt::optional<LightLeSample> SpotLight::sample_le(const Point2f u1, const Point2f u2,
                                                   SampledWavelengths &lambda) const {
    // Choose whether to sample spotlight center cone or falloff region
    FloatType p[2] = {1 - cosFalloffStart, (cosFalloffStart - cosFalloffEnd) / 2};
    FloatType sectionPDF;
    int section = sample_discrete(p, 2, u2[0], &sectionPDF);

    // Sample chosen region of spotlight cone
    Vector3f wLight;
    FloatType pdfDir;
    if (section == 0) {
        // Sample spotlight center cone
        wLight = SampleUniformCone(u1, cosFalloffStart);
        pdfDir = UniformConePDF(cosFalloffStart) * sectionPDF;

    } else {
        // Sample spotlight falloff region
        FloatType cosTheta = SampleSmoothStep(u1[0], cosFalloffEnd, cosFalloffStart);

        FloatType sinTheta = safe_sqrt(1 - sqr(cosTheta));
        FloatType phi = u1[1] * 2 * compute_pi();
        wLight = SphericalDirection(sinTheta, cosTheta, phi);
        pdfDir = SmoothStepPDF(cosTheta, cosFalloffEnd, cosFalloffStart) * sectionPDF /
                 (2 * compute_pi());
    }

    // Return sampled spotlight ray
    auto ray = render_from_light(Ray(Point3f(0, 0, 0), wLight));

    return LightLeSample(I(wLight, lambda), ray, 1, pdfDir);
}

PBRT_CPU_GPU
void SpotLight::pdf_le(const Ray &ray, FloatType *pdfPos, FloatType *pdfDir) const {
    FloatType p[2] = {1 - cosFalloffStart, (cosFalloffStart - cosFalloffEnd) / 2};
    *pdfPos = 0;
    // Find spotlight directional PDF based on $\cos \theta$
    FloatType cosTheta = render_from_light.apply_inverse(ray.d).cos_theta();
    if (cosTheta >= cosFalloffStart) {
        *pdfDir = UniformConePDF(cosFalloffStart) * p[0] / (p[0] + p[1]);
    } else {
        *pdfDir = SmoothStepPDF(cosTheta, cosFalloffEnd, cosFalloffStart) * p[1] /
                  ((p[0] + p[1]) * (2 * compute_pi()));
    }
}

PBRT_CPU_GPU
FloatType SpotLight::pdf_li(const LightSampleContext &ctx, const Vector3f &wi,
                            bool allow_incomplete_pdf) const {
    return 0.0;
}

PBRT_CPU_GPU
SampledSpectrum SpotLight::phi(const SampledWavelengths &lambda) const {
    return scale * i_emit->sample(lambda) * 2 * compute_pi() *
           ((1 - cosFalloffStart) + (cosFalloffStart - cosFalloffEnd) / 2);
}

PBRT_CPU_GPU
SampledSpectrum SpotLight::I(const Vector3f &w, const SampledWavelengths &lambda) const {
    return smooth_step(w.cos_theta(), cosFalloffEnd, cosFalloffStart) * scale *
           i_emit->sample(lambda);
}
