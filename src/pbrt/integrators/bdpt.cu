#include "hip/hip_runtime.h"
#include <pbrt/accelerator/hlbvh.h>
#include <pbrt/base/film.h>
#include <pbrt/base/integrator_base.h>
#include <pbrt/base/interaction.h>
#include <pbrt/base/material.h>
#include <pbrt/base/sampler.h>
#include <pbrt/gpu/gpu_memory_allocator.h>
#include <pbrt/gui/gl_helper.h>
#include <pbrt/integrators/bdpt.h>
#include <pbrt/light_samplers/power_light_sampler.h>
#include <pbrt/lights/image_infinite_light.h>
#include <pbrt/samplers/independent.h>
#include <pbrt/samplers/stratified.h>
#include <pbrt/scene/parameter_dictionary.h>

constexpr size_t NUM_SAMPLERS = 64 * 1024;

struct BDPTSample {
    Point2i p_pixel;
    FloatType weight;
    SampledSpectrum l_path;
    SampledWavelengths lambda;
};

struct FilmSample {
    Point2f p_film;
    SampledSpectrum l_path;
    SampledWavelengths lambda;

    // to help sorting
    bool operator<(const FilmSample &right) const {
        if (p_film.x < right.p_film.x) {
            return true;
        }
        if (p_film.x > right.p_film.x) {
            return false;
        }

        if (p_film.y < right.p_film.y) {
            return true;
        }
        if (p_film.y > right.p_film.y) {
            return false;
        }

        for (int idx = 0; idx < NSpectrumSamples; ++idx) {
            if (l_path[idx] < right.l_path[idx]) {
                return true;
            }
            if (l_path[idx] > right.l_path[idx]) {
                return false;
            }

            if (lambda[idx] < right.lambda[idx]) {
                return true;
            }
            if (lambda[idx] > right.lambda[idx]) {
                return false;
            }
        }

        return false;
    }
};

static __global__ void gpu_init_stratified_samplers(Sampler *samplers,
                                                    StratifiedSampler *stratified_samplers,
                                                    uint samples_per_dimension, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    stratified_samplers[worker_idx].init(samples_per_dimension);

    samplers[worker_idx].init(&stratified_samplers[worker_idx]);
}

static __global__ void gpu_init_independent_samplers(Sampler *samplers,
                                                     IndependentSampler *independent_samplers,
                                                     uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    samplers[worker_idx].init(&independent_samplers[worker_idx]);
}

enum class VertexType { camera, light, surface };

template <typename Type>
class ScopedAssignment {
  public:
    PBRT_CPU_GPU
    explicit ScopedAssignment(Type *_target = nullptr, Type value = Type()) : target(_target) {
        if (_target) {
            backup = *_target;
            *_target = value;
        }
    }

    PBRT_CPU_GPU
    void assign() {
        if (target)
            *target = backup;
    }

    PBRT_CPU_GPU
    ScopedAssignment &operator=(ScopedAssignment &&other) {
        target = other.target;
        backup = other.backup;
        other.target = nullptr;
        return *this;
    }

  private:
    Type *target;
    Type backup;
};

struct EndpointInteraction : Interaction {
    const Camera *camera;
    const Light *light;

    PBRT_CPU_GPU
    EndpointInteraction() : Interaction(), camera(nullptr), light(nullptr) {}

    PBRT_CPU_GPU
    EndpointInteraction(const Light *light, const Ray &r)
        : Interaction(r.o), camera(nullptr), light(light) {}

    PBRT_CPU_GPU
    EndpointInteraction(const Camera *camera, const Ray &ray)
        : Interaction(ray.o), camera(camera), light(nullptr) {}

    PBRT_CPU_GPU
    EndpointInteraction(const Light *light, const Interaction &intr)
        : Interaction(intr), camera(nullptr), light(light) {}

    PBRT_CPU_GPU
    EndpointInteraction(const Interaction &it, const Camera *camera)
        : Interaction(it), camera(camera), light(nullptr) {}

    PBRT_CPU_GPU
    EndpointInteraction(const Ray &ray)
        : Interaction(ray.at(1), Normal3f(-ray.d)), camera(nullptr), light(nullptr) {}
};

PBRT_CPU_GPU
FloatType infinite_light_density(const Light **infinite_lights, int num_infinite_lights,
                                 const PowerLightSampler *lightSampler, const Vector3f w) {
    FloatType pdf = 0;
    for (auto idx = 0; idx < num_infinite_lights; ++idx) {
        auto light = infinite_lights[idx];
        pdf += light->pdf_li(LightSampleContext(Interaction()), -w) * lightSampler->pmf(light);
    }

    return pdf;
}

struct Vertex {
    VertexType type;
    SampledSpectrum beta;
    EndpointInteraction ei;
    SurfaceInteraction si;
    BSDF bsdf;

    bool delta;
    FloatType pdfFwd;
    FloatType pdfRev;

    PBRT_CPU_GPU
    Vertex() : type(VertexType::camera), beta(NAN), delta(false), pdfFwd(0), pdfRev(0) {}

    PBRT_CPU_GPU
    Vertex(VertexType _type, const EndpointInteraction &_ei, const SampledSpectrum &_beta)
        : type(_type), beta(_beta), delta(false), pdfFwd(0), pdfRev(0), ei(_ei) {}

    PBRT_CPU_GPU
    Vertex(const SurfaceInteraction &_si, const BSDF &_bsdf, const SampledSpectrum &_beta)
        : type(VertexType::surface), beta(_beta), delta(false), pdfFwd(0), pdfRev(0), si(_si),
          bsdf(_bsdf) {}

    PBRT_CPU_GPU
    bool is_light() const {
        return type == VertexType::light ||
               (type == VertexType::surface && si.area_light != nullptr);
    }

    PBRT_CPU_GPU
    bool is_delta_light() const {
        return type == VertexType::light && ei.light &&
               pbrt::is_delta_light(ei.light->get_light_type());
    }

    PBRT_CPU_GPU
    static Vertex create_camera(const Camera *camera, const Ray &ray, const SampledSpectrum &beta) {
        return Vertex(VertexType::camera, EndpointInteraction(camera, ray), beta);
    }

    PBRT_CPU_GPU
    static Vertex create_camera(const Camera *camera, const Interaction &it,
                                const SampledSpectrum &beta) {
        return Vertex(VertexType::camera, EndpointInteraction(it, camera), beta);
    }

    PBRT_CPU_GPU
    static Vertex create_light(const EndpointInteraction &ei, const SampledSpectrum &beta,
                               FloatType pdf) {
        Vertex v(VertexType::light, ei, beta);
        v.pdfFwd = pdf;
        return v;
    }

    PBRT_CPU_GPU
    static Vertex create_light(const Light *light, const Interaction &intr,
                               const SampledSpectrum &Le, FloatType pdf) {
        Vertex v(VertexType::light, EndpointInteraction(light, intr), Le);
        v.pdfFwd = pdf;
        return v;
    }

    PBRT_CPU_GPU
    static Vertex create_light(const Light *light, const Ray &ray, const SampledSpectrum &Le,
                               FloatType pdf) {
        Vertex v(VertexType::light, EndpointInteraction(light, ray), Le);
        v.pdfFwd = pdf;
        return v;
    }

    PBRT_CPU_GPU
    static Vertex create_surface(const SurfaceInteraction &si, const BSDF &bsdf,
                                 const SampledSpectrum &beta, FloatType pdf, const Vertex &prev) {
        Vertex v(si, bsdf, beta);
        v.pdfFwd = prev.convert_density(pdf, v);
        return v;
    }

    PBRT_CPU_GPU
    bool is_connectible() const {
        switch (type) {
        case VertexType::light: {
            return ei.light->get_light_type() != LightType::delta_direction;
        }
        case VertexType::camera: {
            return true;
        }
        case VertexType::surface: {
            return pbrt::is_non_specular(bsdf.flags());
        }
        }

        REPORT_FATAL_ERROR();
        return false;
    }

    PBRT_CPU_GPU
    const Interaction &get_interaction() const {
        switch (type) {
        case VertexType::surface: {
            return si;
        }
        default: {
            return ei;
        }
        }

        REPORT_FATAL_ERROR();
    }

    PBRT_CPU_GPU
    const SurfaceInteraction &get_surface_interaction() const {
        if (type == VertexType::surface) {
            return si;
        }

        REPORT_FATAL_ERROR();
        SurfaceInteraction unused;
        return unused;
    }

    PBRT_CPU_GPU
    Point3f p() const {
        return get_interaction().p();
    }

    PBRT_CPU_GPU
    const Normal3f &ng() const {
        return get_interaction().n;
    }

    PBRT_CPU_GPU
    const Normal3f &ns() const {
        if (type == VertexType::surface) {
            return si.shading.n;
        }

        return get_interaction().n;
    }

    PBRT_CPU_GPU
    bool is_on_surface() const {
        return get_interaction().is_surface_interaction();
    }

    PBRT_CPU_GPU
    SampledSpectrum f(const Vertex &next, TransportMode mode) const {
        Vector3f wi = next.p() - p();

        if (wi.squared_length() == 0) {
            return SampledSpectrum(0);
        }

        wi = wi.normalize();
        switch (type) {
        case VertexType::surface:
            return bsdf.f(si.wo, wi, mode);
        }

        REPORT_FATAL_ERROR();
        return SampledSpectrum(NAN);
    }

    PBRT_CPU_GPU
    bool is_infinite_light() const {
        return type == VertexType::light &&
               (!ei.light || ei.light->get_light_type() == LightType::infinite ||
                ei.light->get_light_type() == LightType::delta_direction);
    }

    PBRT_CPU_GPU
    FloatType convert_density(FloatType pdf, const Vertex &next) const {
        // Return solid angle density if _next_ is an infinite area light
        if (next.is_infinite_light()) {
            return pdf;
        }

        Vector3f w = next.p() - p();
        if (w.squared_length() == 0) {
            return 0;
        }

        FloatType invDist2 = 1 / w.squared_length();
        if (next.is_on_surface()) {
            pdf *= next.ng().abs_dot(w * std::sqrt(invDist2));
        }

        return pdf * invDist2;
    }

    PBRT_CPU_GPU
    FloatType pdf_light(const IntegratorBase *integrator_base, const Vertex &v) const {
        Vector3f w = v.p() - p();
        auto invDist2 = 1.0 / w.squared_length();
        w *= std::sqrt(invDist2);

        // Compute sampling density _pdf_ for light type
        FloatType pdf;
        if (is_infinite_light()) {
            // Compute planar sampling density for infinite light sources
            Bounds3f sceneBounds = integrator_base->bvh->bounds();
            Point3f sceneCenter;
            FloatType sceneRadius;
            sceneBounds.bounding_sphere(&sceneCenter, &sceneRadius);
            pdf = 1.0 / (compute_pi() * sqr(sceneRadius));
        } else if (is_on_surface()) {
            // Compute sampling density at emissive surface
            if constexpr (DEBUG_MODE && type == VertexType::light) {
                if (ei.light->get_light_type() != LightType::area) {
                    REPORT_FATAL_ERROR();
                }
            }

            auto light = (type == VertexType::light) ? ei.light : si.area_light;
            FloatType pdfPos, pdfDir;
            light->pdf_le(ei, w, &pdfPos, &pdfDir);
            pdf = pdfDir * invDist2;
        } else {
            if constexpr (DEBUG_MODE) {
                if (type != VertexType::light || ei.light == nullptr) {
                    REPORT_FATAL_ERROR();
                }
            }

            // Compute sampling density for noninfinite light sources
            FloatType pdfPos, pdfDir;
            ei.light->pdf_le(Ray(p(), w), &pdfPos, &pdfDir);
            pdf = pdfDir * invDist2;
        }

        if (v.is_on_surface()) {
            pdf *= v.ng().abs_dot(w);
        }

        return pdf;
    }

    PBRT_CPU_GPU
    FloatType pdf(const IntegratorBase *integrator_base, const Vertex *prev,
                  const Vertex &next) const {
        if (type == VertexType::light) {
            return pdf_light(integrator_base, next);
        }

        // Compute directions to preceding and next vertex
        Vector3f wn = next.p() - p();
        if (wn.squared_length() == 0) {
            return 0;
        }

        wn = wn.normalize();
        Vector3f wp;
        if (prev) {
            wp = prev->p() - p();
            if (wp.squared_length() == 0) {
                return 0;
            }
            wp = wp.normalize();
        } else {
            if constexpr (DEBUG_MODE && type != VertexType::camera) {
                REPORT_FATAL_ERROR();
            }
        }

        // Compute directional density depending on the vertex type
        FloatType pdf = 0;

        switch (type) {
        case VertexType::camera: {
            FloatType unused;
            ei.camera->pdf_we(ei.spawn_ray(wn), &unused, &pdf);
            break;
        }
        case VertexType::surface: {
            pdf = bsdf.pdf(wp, wn);
            break;
        }
        default: {
            REPORT_FATAL_ERROR();
        }
        }

        // Return probability per unit area at vertex _next_
        return convert_density(pdf, next);
    }

    PBRT_CPU_GPU
    SampledSpectrum Le(const Light **infinite_lights, int num_infinite_lights, const Vertex &v,
                       const SampledWavelengths &lambda) const {
        if (!is_light()) {
            return SampledSpectrum(0.0);
        }

        Vector3f w = v.p() - p();
        if (w.squared_length() == 0) {
            return SampledSpectrum(0.0);
        }

        w = w.normalize();
        if (is_infinite_light()) {
            // Return emitted radiance for infinite light sources
            SampledSpectrum Le(0.f);

            for (uint idx = 0; idx < num_infinite_lights; ++idx) {
                auto light = infinite_lights[idx];
                Le += light->le(Ray(p(), -w), lambda);
            }

            return Le;
        }

        if (si.area_light != nullptr) {
            return si.area_light->l(si.p(), si.n, si.uv, w, lambda);
        }

        return SampledSpectrum(0.f);
    }

    PBRT_CPU_GPU
    FloatType pdf_light_origin(const Light **infinite_lights, int num_infinite_lights,
                               const Vertex &v, const PowerLightSampler *lightSampler) {
        Vector3f w = v.p() - p();
        if (w.squared_length() == 0) {
            return 0.0;
        }

        w = w.normalize();

        if (is_infinite_light()) {
            // Return sampling density for infinite light sources
            return infinite_light_density(infinite_lights, num_infinite_lights, lightSampler, w);
        }

        // Return sampling density for noninfinite light source
        auto light = type == VertexType::light ? ei.light : si.area_light;

        FloatType pdfPos, pdfDir;
        auto pdfChoice = lightSampler->pmf(light);

        if (is_on_surface()) {
            light->pdf_le(ei, w, &pdfPos, &pdfDir);
        } else {
            light->pdf_le(Ray(p(), w), &pdfPos, &pdfDir);
        }

        return pdfPos * pdfChoice;
    }
};

PBRT_CPU_GPU
SampledSpectrum G(const IntegratorBase *integrator_base, const Vertex &v0, const Vertex &v1,
                  const SampledWavelengths &lambda) {
    Vector3f d = v0.p() - v1.p();
    auto g = 1.0 / d.squared_length();
    d *= std::sqrt(g);
    if (v0.is_on_surface()) {
        g *= v0.ns().abs_dot(d);
    }

    if (v1.is_on_surface()) {
        g *= v1.ns().abs_dot(d);
    }

    return g * integrator_base->tr(v0.get_interaction(), v1.get_interaction());
}

PBRT_CPU_GPU
FloatType mis_weight(const IntegratorBase *integrator_base, Vertex *lightVertices,
                     Vertex *cameraVertices, Vertex &sampled, int s, int t) {
    if (s + t == 2) {
        return 1;
    }

    // Define helper function _remap0_ that deals with Dirac delta functions
    auto remap0 = [](float f) -> FloatType { return f != 0 ? f : 1.0; };

    // Temporarily update vertex properties for current strategy
    // Look up connection vertices and their predecessors
    Vertex *qs = s > 0 ? &lightVertices[s - 1] : nullptr,
           *pt = t > 0 ? &cameraVertices[t - 1] : nullptr,
           *qsMinus = s > 1 ? &lightVertices[s - 2] : nullptr,
           *ptMinus = t > 1 ? &cameraVertices[t - 2] : nullptr;

    // Update sampled vertex for $s=1$ or $t=1$ strategy
    ScopedAssignment<Vertex> a1;
    if (s == 1) {
        a1 = ScopedAssignment(qs, sampled);
    } else if (t == 1) {
        a1 = ScopedAssignment(pt, sampled);
    }

    // Mark connection vertices as non-degenerate
    ScopedAssignment<bool> a2, a3;
    if (pt) {
        a2 = ScopedAssignment(&pt->delta, false);
    }
    if (qs) {
        a3 = ScopedAssignment(&qs->delta, false);
    }

    // Update reverse density of vertex $\pt{}_{t-1}$
    ScopedAssignment<FloatType> a4;
    if (pt) {
        a4 = ScopedAssignment(
            &pt->pdfRev, s > 0 ? qs->pdf(integrator_base, qsMinus, *pt)
                               : pt->pdf_light_origin(integrator_base->infinite_lights,
                                                      integrator_base->infinite_light_num, *ptMinus,
                                                      integrator_base->light_sampler));
    }

    // Update reverse density of vertex $\pt{}_{t-2}$
    ScopedAssignment<FloatType> a5;
    if (ptMinus) {
        a5 = ScopedAssignment(&ptMinus->pdfRev, s > 0 ? pt->pdf(integrator_base, qs, *ptMinus)
                                                      : pt->pdf_light(integrator_base, *ptMinus));
    }

    // Update reverse density of vertices $\pq{}_{s-1}$ and $\pq{}_{s-2}$
    ScopedAssignment<FloatType> a6;
    if (qs) {
        a6 = ScopedAssignment(&qs->pdfRev, pt->pdf(integrator_base, ptMinus, *qs));
    }

    ScopedAssignment<FloatType> a7;
    if (qsMinus) {
        a7 = ScopedAssignment(&qsMinus->pdfRev, qs->pdf(integrator_base, pt, *qsMinus));
    }

    FloatType sumRi = 0;

    // Consider hypothetical connection strategies along the camera subpath
    FloatType ri = 1.0;
    for (int i = t - 1; i > 0; --i) {
        ri *= remap0(cameraVertices[i].pdfRev) / remap0(cameraVertices[i].pdfFwd);
        if (!cameraVertices[i].delta && !cameraVertices[i - 1].delta) {
            sumRi += ri;
        }
    }

    ri = 1;
    for (int i = s - 1; i >= 0; --i) {
        ri *= remap0(lightVertices[i].pdfRev) / remap0(lightVertices[i].pdfFwd);
        bool deltaLightvertex =
            i > 0 ? lightVertices[i - 1].delta : lightVertices[0].is_delta_light();
        if (!lightVertices[i].delta && !deltaLightvertex) {
            sumRi += ri;
        }
    }

    a1.assign();
    a2.assign();
    a3.assign();
    a4.assign();
    a5.assign();
    a6.assign();
    a7.assign();

    return 1.0 / (1.0 + sumRi);
}

PBRT_CPU_GPU
int random_walk(const IntegratorBase *integrator_base, SampledWavelengths &lambda, Ray ray,
                Sampler *sampler, SampledSpectrum beta, FloatType pdf, int maxDepth,
                TransportMode mode, Vertex *path, bool regularize) {
    if (maxDepth == 0) {
        return 0;
    }

    const auto camera = integrator_base->camera;

    // Follow random walk to initialize BDPT path vertices
    int bounces = 0;
    bool anyNonSpecularBounces = false;
    auto pdfFwd = pdf;
    while (true) {
        if (!beta.is_positive()) {
            break;
        }

        bool scattered = false;
        bool terminated = false;

        // Trace a ray and sample the medium, if any
        Vertex &vertex = path[bounces];
        Vertex &prev = path[bounces - 1];
        auto si = integrator_base->intersect(ray, Infinity);

        if (terminated) {
            return bounces;
        }
        if (scattered) {
            continue;
        }

        // Handle escaped rays after no medium scattering event
        if (!si) {
            // Capture escaped rays when tracing from the camera
            if (mode == TransportMode::Radiance) {
                vertex = Vertex::create_light(EndpointInteraction(ray), beta, pdfFwd);
                ++bounces;
            }
            break;
        }

        // Handle surface interaction for path generation
        SurfaceInteraction &isect = si->interaction;
        // Get BSDF and skip over medium boundaries

        auto bsdf = isect.get_bsdf(lambda, camera, sampler->get_samples_per_pixel());

        // Possibly regularize the BSDF
        if (regularize && anyNonSpecularBounces) {
            bsdf.regularize();
        }

        // Initialize _vertex_ with surface intersection information
        vertex = Vertex::create_surface(isect, bsdf, beta, pdfFwd, prev);

        if (++bounces >= maxDepth) {
            break;
        }

        // Sample BSDF at current vertex
        Vector3f wo = isect.wo;
        auto u = sampler->get_1d();

        auto bs = vertex.bsdf.sample_f(wo, u, sampler->get_2d(), mode);
        if (!bs) {
            break;
        }

        pdfFwd = bs->pdf_is_proportional ? vertex.bsdf.pdf(wo, bs->wi, mode) : bs->pdf;
        anyNonSpecularBounces |= !bs->is_specular();

        beta *= bs->f * isect.shading.n.abs_dot(bs->wi) / bs->pdf;
        ray = isect.spawn_ray(bs->wi);
        // spawn_ray() is simplified from the original one from PBRT-v4

        auto _pdfRev = vertex.bsdf.pdf(bs->wi, wo, !mode);

        if (bs->is_specular()) {
            vertex.delta = true;
            _pdfRev = pdfFwd = 0;
        }

        prev.pdfRev = vertex.convert_density(_pdfRev, prev);
    }

    return bounces;
}

PBRT_CPU_GPU
int generate_camera_subpath(const IntegratorBase *integrator_base, const Ray &ray,
                            SampledWavelengths &lambda, Sampler *sampler, int maxDepth,
                            Vertex *path, bool regularize) {
    if (maxDepth == 0) {
        return 0;
    }

    const auto camera = integrator_base->camera;

    SampledSpectrum beta(1.f);
    // Generate first vertex on camera subpath and start random walk
    FloatType pdfPos, pdfDir;

    path[0] = Vertex::create_camera(camera, ray, beta);

    camera->pdf_we(ray, &pdfPos, &pdfDir);

    return random_walk(integrator_base, lambda, ray, sampler, beta, pdfDir, maxDepth - 1,
                       TransportMode::Radiance, path + 1, regularize) +
           1;
}

PBRT_CPU_GPU
int generate_light_subpath(const IntegratorBase *integrator_base, SampledWavelengths &lambda,
                           Sampler *sampler, int maxDepth, Vertex *path, bool regularize) {
    // Generate light subpath and initialize _path_ vertices
    if (maxDepth == 0) {
        return 0;
    }

    // Sample initial ray for light subpath
    // Sample light for BDPT light subpath
    auto sampledLight = integrator_base->light_sampler->sample(sampler->get_1d());
    if (!sampledLight) {
        return 0;
    }

    auto light = sampledLight->light;
    auto lightSamplePDF = sampledLight->p;

    auto ul0 = sampler->get_2d();
    auto ul1 = sampler->get_2d();
    auto les = light->sample_le(ul0, ul1, lambda);

    if (!les || les->pdfPos == 0 || les->pdfDir == 0 || !les->L.is_positive()) {
        return 0;
    }

    auto ray = les->ray;

    // Generate first vertex of light subpath
    auto p_l = lightSamplePDF * les->pdfPos;
    path[0] = les->intr ? Vertex::create_light(light, *les->intr, les->L, p_l)
                        : Vertex::create_light(light, ray, les->L, p_l);

    // Follow light subpath random walk
    SampledSpectrum beta = les->L * les->abs_cos_theta(ray.d) / (p_l * les->pdfDir);

    int nVertices = random_walk(integrator_base, lambda, ray, sampler, beta, les->pdfDir,
                                maxDepth - 1, TransportMode::Importance, path + 1, regularize);

    // Correct subpath sampling densities for infinite area lights
    if (path[0].is_infinite_light()) {
        // Set spatial density of _path[1]_ for infinite area light
        if (nVertices > 0) {
            path[1].pdfFwd = les->pdfPos;
            if (path[1].is_on_surface()) {
                path[1].pdfFwd *= path[1].ng().abs_dot(ray.d);
            }
        }

        // Set spatial density of _path[0]_ for infinite area light
        path[0].pdfFwd = infinite_light_density(integrator_base->infinite_lights,
                                                integrator_base->infinite_light_num,
                                                integrator_base->light_sampler, ray.d);
    }

    return nVertices + 1;
}

PBRT_CPU_GPU
SampledSpectrum connect_bdpt(const IntegratorBase *integrator_base, SampledWavelengths &lambda,
                             Vertex *lightVertices, Vertex *cameraVertices, int s, int t,
                             Sampler *sampler, pbrt::optional<Point2f> *pRaster,
                             FloatType *misWeightPtr = nullptr) {
    SampledSpectrum L(0.f);
    // Ignore invalid connections related to infinite area lights
    if (t > 1 && s != 0 && cameraVertices[t - 1].type == VertexType::light) {
        return SampledSpectrum(0);
    }

    auto camera = integrator_base->camera;

    // Perform connection and write contribution to _L_
    Vertex sampled;
    if (s == 0) {
        // Interpret the camera subpath as a complete path
        const Vertex &pt = cameraVertices[t - 1];
        if (pt.is_light()) {
            L = pt.Le(integrator_base->infinite_lights, integrator_base->infinite_light_num,
                      cameraVertices[t - 2], lambda) *
                pt.beta;
        }

    } else if (t == 1) {
        // Sample a point on the camera and connect it to the light subpath
        const Vertex &qs = lightVertices[s - 1];
        if (qs.is_connectible()) {
            auto cs = camera->sample_wi(qs.get_interaction(), sampler->get_2d(), lambda);
            if (cs) {
                *pRaster = cs->pRaster;
                // Initialize dynamically sampled vertex and _L_ for $t=1$ case
                sampled = Vertex::create_camera(camera, cs->pLens, cs->Wi / cs->pdf);

                L = qs.beta * qs.f(sampled, TransportMode::Importance) * sampled.beta;
                if (qs.is_on_surface()) {
                    L *= qs.ns().abs_dot(cs->wi);
                }

                if (L.is_positive()) {
                    L *= integrator_base->tr(cs->pRef, cs->pLens);
                }
            }
        }
    } else if (s == 1) {
        // Sample a point on a light and connect it to the camera subpath
        const Vertex &pt = cameraVertices[t - 1];
        if (pt.is_connectible()) {
            auto sampledLight = integrator_base->light_sampler->sample(sampler->get_1d());
            if (sampledLight) {
                auto light = sampledLight->light;
                auto p_l = sampledLight->p;

                LightSampleContext ctx;
                if (pt.is_on_surface()) {
                    const SurfaceInteraction &si = pt.get_surface_interaction();
                    ctx = LightSampleContext(si);
                    // Try to nudge the light sampling position to correct side of the
                    // surface
                    BxDFFlags flags = pt.bsdf.flags();
                    if (pbrt::is_reflective(flags) && !pbrt::is_transmissive(flags)) {
                        ctx.pi = si.offset_ray_origin(si.wo);
                    } else if (pbrt::is_transmissive(flags) && !pbrt::is_reflective(flags)) {
                        ctx.pi = si.offset_ray_origin(-si.wo);
                    }
                } else {
                    ctx = LightSampleContext(pt.get_interaction());
                }

                auto lightWeight = light->sample_li(ctx, sampler->get_2d(), lambda);
                if (lightWeight && lightWeight->l.is_positive() && lightWeight->pdf > 0) {
                    EndpointInteraction ei(light, lightWeight->p_light);

                    sampled =
                        Vertex::create_light(ei, lightWeight->l / (lightWeight->pdf * p_l), 0);
                    sampled.pdfFwd = sampled.pdf_light_origin(integrator_base->infinite_lights,
                                                              integrator_base->infinite_light_num,
                                                              pt, integrator_base->light_sampler);

                    L = pt.beta * pt.f(sampled, TransportMode::Radiance) * sampled.beta;

                    if (pt.is_on_surface()) {
                        L *= pt.ns().abs_dot(lightWeight->wi);
                    }

                    // Only check visibility if the path would carry radiance.
                    if (L.is_positive()) {
                        L *= integrator_base->tr(pt.get_interaction(), lightWeight->p_light);
                    }
                }
            }
        }
    } else {
        // Handle all other bidirectional connection cases
        const Vertex &qs = lightVertices[s - 1], &pt = cameraVertices[t - 1];
        if (qs.is_connectible() && pt.is_connectible()) {
            L = qs.beta * qs.f(pt, TransportMode::Importance) * pt.f(qs, TransportMode::Radiance) *
                pt.beta;

            if (L.is_positive()) {
                L *= G(integrator_base, qs, pt, lambda);
            }
        }
    }

    // Compute MIS weight for connection strategy
    FloatType misWeight =
        L.is_positive() ? mis_weight(integrator_base, lightVertices, cameraVertices, sampled, s, t)
                        : 0.0;

    L *= misWeight;
    if (misWeightPtr != nullptr) {
        *misWeightPtr = misWeight;
    }

    return L;
}

BDPTIntegrator *BDPTIntegrator::create(int samples_per_pixel, const std::string &sampler_type,
                                       const ParameterDictionary &parameters,
                                       const IntegratorBase *integrator_base,
                                       GPUMemoryAllocator &allocator) {
    auto bdpt_integrator = allocator.allocate<BDPTIntegrator>();
    auto samplers = allocator.allocate<Sampler>(NUM_SAMPLERS);

    bdpt_integrator->samplers = samplers;
    bdpt_integrator->base = integrator_base;
    bdpt_integrator->max_depth = parameters.get_integer("maxdepth", 10);
    bdpt_integrator->regularize = parameters.get_bool("regularize", false);
    bdpt_integrator->film_sample_size = bdpt_integrator->max_depth * NUM_SAMPLERS;

    const uint threads = 1024;
    uint blocks = divide_and_ceil<uint>(NUM_SAMPLERS, threads);

    if (sampler_type == "independent") {
        auto independent_samplers = allocator.allocate<IndependentSampler>(NUM_SAMPLERS);

        gpu_init_independent_samplers<<<blocks, threads>>>(samplers, independent_samplers,
                                                           NUM_SAMPLERS);
    } else if (sampler_type == "stratified") {
        const uint samples_per_dimension = std::sqrt(samples_per_pixel);
        if (samples_per_dimension * samples_per_dimension != samples_per_pixel) {
            REPORT_FATAL_ERROR();
        }

        auto stratified_samplers = allocator.allocate<StratifiedSampler>(NUM_SAMPLERS);

        gpu_init_stratified_samplers<<<blocks, threads>>>(samplers, stratified_samplers,
                                                          samples_per_dimension, NUM_SAMPLERS);
    } else {
        REPORT_FATAL_ERROR();
    }

    return bdpt_integrator;
}

__global__ void wavefront_render(BDPTSample *bdpt_samples, FilmSample *film_samples,
                                 int *film_sample_counter, Vertex *global_camera_vertices,
                                 Vertex *global_light_vertices, uint pass, uint samples_per_pixel,
                                 const Point2i film_resolution, BDPTIntegrator *bdpt_integrator) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= NUM_SAMPLERS) {
        return;
    }

    const auto width = film_resolution.x;
    const auto height = film_resolution.y;

    auto global_idx = (long long)(pass)*NUM_SAMPLERS + worker_idx;

    const auto pixel_idx = global_idx % (width * height);
    const auto sample_idx = global_idx / (width * height);
    if (sample_idx >= samples_per_pixel) {
        return;
    }

    auto local_sampler = &bdpt_integrator->samplers[worker_idx];
    local_sampler->start_pixel_sample(pixel_idx, sample_idx, 0);

    auto p_pixel = Point2i(pixel_idx % width, pixel_idx / width);

    auto camera_sample = local_sampler->get_camera_sample(p_pixel, bdpt_integrator->base->filter);

    auto lu = local_sampler->get_1d();
    auto lambda = SampledWavelengths::sample_visible(lu);

    auto ray = bdpt_integrator->base->camera->generate_ray(camera_sample, local_sampler);

    auto local_camera_vertices =
        &global_camera_vertices[worker_idx * (bdpt_integrator->max_depth + 2)];
    auto local_light_vertices =
        &global_light_vertices[worker_idx * (bdpt_integrator->max_depth + 1)];

    auto rendered_sample = &bdpt_samples[worker_idx];

    auto radiance_l = ray.weight * bdpt_integrator->li(film_samples, film_sample_counter, ray.ray,
                                                       lambda, local_sampler, local_camera_vertices,
                                                       local_light_vertices);

    rendered_sample->p_pixel = p_pixel;
    rendered_sample->weight = camera_sample.filter_weight;
    rendered_sample->l_path = radiance_l;
    rendered_sample->lambda = lambda;
}

void BDPTIntegrator::render(Film *film, uint samples_per_pixel, const bool preview) {
    const auto image_resolution = film->get_resolution();

    GPUMemoryAllocator local_allocator;

    GLHelper gl_helper;
    if (preview) {
        gl_helper.init("initializing", image_resolution);
    }

    auto bdpt_samples = local_allocator.allocate<BDPTSample>(NUM_SAMPLERS);

    auto film_samples = local_allocator.allocate<FilmSample>(film_sample_size);

    auto film_sample_counter = local_allocator.allocate<int>();

    auto global_camera_vertices = local_allocator.allocate<Vertex>(NUM_SAMPLERS * (max_depth + 2));
    auto global_light_vertices = local_allocator.allocate<Vertex>(NUM_SAMPLERS * (max_depth + 1));

    auto num_pixels = image_resolution.x * image_resolution.y;

    constexpr uint threads = 32;
    const uint blocks = divide_and_ceil<uint>(NUM_SAMPLERS, threads);

    auto total_pass = divide_and_ceil<long long>(num_pixels * samples_per_pixel, NUM_SAMPLERS);

    for (uint pass = 0; pass < total_pass; ++pass) {
        *film_sample_counter = 0;
        wavefront_render<<<blocks, threads>>>(bdpt_samples, film_samples, film_sample_counter,
                                              global_camera_vertices, global_light_vertices, pass,
                                              samples_per_pixel, film->get_resolution(), this);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        for (uint idx = 0; idx < NUM_SAMPLERS; ++idx) {
            const auto global_idx = (long long)(pass)*NUM_SAMPLERS + idx;
            const auto sample_idx = global_idx / num_pixels;
            if (sample_idx >= samples_per_pixel) {
                return;
            }

            const auto sample = &bdpt_samples[idx];
            film->add_sample(sample->p_pixel, sample->l_path, sample->lambda, sample->weight);
        }

        if (*film_sample_counter > 0) {
            // sort to make film writing deterministic
            std::sort(film_samples + 0, film_samples + (*film_sample_counter), std::less{});
        }

        for (uint idx = 0; idx < *film_sample_counter; ++idx) {
            const auto sample = &film_samples[idx];
            film->add_splat(sample->p_film, sample->l_path, sample->lambda);
        }

        if (preview) {
            film->copy_to_frame_buffer(gl_helper.gpu_frame_buffer, 1.0 / samples_per_pixel);
            gl_helper.draw_frame(GLHelper::assemble_title(FloatType(pass + 1) / total_pass));
        }
    }
}

PBRT_GPU
SampledSpectrum BDPTIntegrator::li(FilmSample *film_samples, int *film_sample_counter,
                                   const Ray &ray, SampledWavelengths &lambda, Sampler *sampler,
                                   Vertex *camera_vertices, Vertex *light_vertices) const {
    // Trace the camera and light subpaths
    int nCamera = generate_camera_subpath(base, ray, lambda, sampler, max_depth + 2,
                                          camera_vertices, regularize);
    int nLight =
        generate_light_subpath(base, lambda, sampler, max_depth + 1, light_vertices, regularize);

    SampledSpectrum accumulated_l(0);

    // Execute all BDPT connection strategies
    for (int t = 1; t <= nCamera; ++t) {
        for (int s = 0; s <= nLight; ++s) {
            int depth = t + s - 2;
            if ((s == 1 && t == 1) || depth < 0 || depth > max_depth) {
                continue;
            }

            // Execute the $(s, t)$ connection strategy and update _L_
            pbrt::optional<Point2f> optional_p_film_new;
            FloatType misWeight = 0;
            SampledSpectrum l_path = connect_bdpt(base, lambda, light_vertices, camera_vertices, s,
                                                  t, sampler, &optional_p_film_new, &misWeight);

            if (t != 1) {
                accumulated_l += l_path;
            } else if (l_path.is_positive()) {
                if constexpr (DEBUG_MODE && !optional_p_film_new.has_value()) {
                    REPORT_FATAL_ERROR();
                }

                const auto film_sample_idx = atomicAdd(film_sample_counter, 1);

                if (film_sample_idx >= film_sample_size) {
                    REPORT_FATAL_ERROR();
                }

                film_samples[film_sample_idx].p_film = optional_p_film_new.value();
                film_samples[film_sample_idx].l_path = l_path;
                film_samples[film_sample_idx].lambda = lambda;
            }
        }
    }

    return accumulated_l;
}
